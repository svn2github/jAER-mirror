#include "hip/hip_runtime.h"
/* jaercuda project for running AER convolution processing on the GPU. 
    From template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code. This file is #included from template.cu.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "config.h"


// this should not be a device pointer, it should be a host pointer which is hipMalloc'ed and which has data copied to it from the host, then bound as texture
// we cannot refer to this data here except by texFetch1d
texture <float> template_tex;
//__device__ float gpu_conv_template[MAX_NUM_TEMPLATE][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE]; // TODO tobi changed to hipMalloc so that memory is accessible, may cause problems

__device__ float gpu_membranePotential[MAX_NUM_TEMPLATE][MAX_Y][MAX_X];
__device__ unsigned long gpu_lastTimeStamp[MAX_NUM_TEMPLATE][MAX_Y][MAX_X];

__constant__ __device__  globalNeuronParams_t constNeuronParams;

__device__ int   numFiring0[MAX_NUM_TEMPLATE];
__device__ int   numFiring1[MAX_NUM_TEMPLATE];  
__device__ unsigned int   firedNeuronAddr[MAX_NUM_TEMPLATE*MAX_FIRING]; // holds output spikes from each template

/** This method implements part of the winner-take-all functionality within each population.
 * The CPU calls this kernel whenever some neurons in the neuron array has fired.
 * Each thread is responsible for inhibition of one neuron potential by iESynWeight amount.
 * Finally each thread clamps the membrane potential to stay within a specific value.
 * We use a ID grid of 128x1 thread and 128*num_object block to have simple addressing mechanism.
 * 2D block of thread can also do similar computation but kernel needs slightly more address calculations.
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	iNeuronFired		each bit records if the global inhibitory neuron of the corresponding excitatory population has fired during the last cycle
 **/
__global__ void WTAKernelMO(int* numFiringAddr, char iNeuronFired) 
{	
	// Thread index
	int my_addrx = threadIdx.x; // this thread handles neuron (x,y)=(threadIdx.x, blockIdx.x)
	int my_addry = blockIdx.x;
	int neuronArrayId = blockIdx.y;
	
	// check if the global inhibitory neuron of a particular population fired
	char iFired = (char)(iNeuronFired & (0x01<<neuronArrayId));

	if(iFired){
		float temp = gpu_membranePotential[neuronArrayId][my_addry][my_addrx]; // membrane potential of an LIF neuron for one template array

		temp -= constNeuronParams.iESynWeight; // reduce it by the iE weight (inhibitory to excitatory)
		
		if ( temp < constNeuronParams.membranePotentialMin )
			// clamp it to negative driving potential (negative weight can never make it fire)
			temp = constNeuronParams.membranePotentialMin; 
			
		gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = temp;
	}		
}

/** This method implements part of the global winner-take-all functionality among populations.
 * The CPU calls this kernel whenever some neurons in the neuron array has fired.
 * Each thread is responsible for inhibition of one neuron potential by iESynWeight amount.
 * Finally each thread clamps the membrane potential to stay within a specific value.
 * We use a ID grid of 128x1 thread and 128*num_object block to have simple addressing mechanism.
 * 2D block of thread can also do similar computation but kernel needs slightly more address calculations.
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	n_iNeuronFired		the number of spikes the global inhibitory neuron fired during the current cycle
 **/
__global__ void WTAKernelMOGlob(int* numFiringAddr, int n_iNeuronFired) 
{	
	// Thread index
	int my_addrx = threadIdx.x; // this thread handles neuron (x,y)=(threadIdx.x, blockIdx.x)
	int my_addry = blockIdx.x;
	int neuronArrayId = blockIdx.y;
	
	// check if the global inhibitory neuron of a particular population fired

	if(n_iNeuronFired != 0){
		float temp = gpu_membranePotential[neuronArrayId][my_addry][my_addrx]; // membrane potential of an LIF neuron for one template array

		temp -= n_iNeuronFired*constNeuronParams.iESynWeight; // reduce it by the iE weight (inhibitory to excitatory)
		
		if ( temp < constNeuronParams.membranePotentialMin )
			// clamp it to negative driving potential (negative weight can never make it fire)
			temp = constNeuronParams.membranePotentialMin; 
			
		gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = temp;
	}		
}

// this array stores the incoming spikes from CPU...
__device__ int gpu_spikeAddr[GPU_MAX_SPIKE_PACKETS];
__device__ unsigned long gpu_spikeTime[GPU_MAX_SPIKE_PACKETS];


/** This kernel is to update the excitatory neurons within each population
 * @param:	numInpSpikes		total number of input spikes within current cycle
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	resetAddr			the array recording the number of spikes generated within each population during the last cycle, needs to be reset during the kernel call
 **/
__global__ void
convNN_multiSpikeKernelNew1(int  numInpSpikes,			// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr)			// This memory will be reset to zero by GPU						
					
{	
	
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = blockIdx.x*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = (blockIdx.y&0x7)*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
	
	//We encode the object dimension in blockId itself.last 3 bit denotes
	//block number, the remaining bits denote the object number neuronArrayId
	int neuronArrayId = (blockIdx.y>>3);
	
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	// only one thread is updated to reduce the global memory access
	if (my_localId == 0) {		
		resetAddr[neuronArrayId]=0; // FAQ: why is the kernel doing this, what is resetAddr??? it's not used further in the kernel
									// we use a simple double buffering scheme. this address will be passed as
									// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
									// by the CPU for reseting the number of firing by this mechanism.
	}
		
   __syncthreads();
   
   __shared__ unsigned long curSpikeTime;
   __shared__ unsigned int curSpikeAddr;

   unsigned long ltStamp = 0;
   // retreive the membrane potential
   ltStamp = gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx];
   
   // retreive the initial value of the membrane potential and multiply by decay value
   float refValue  = gpu_membranePotential[neuronArrayId][my_addry][my_addrx];

   // for each spike from CPU
   for(int spkCnt=0; spkCnt < numInpSpikes;spkCnt++) {

		// only one thread within a block does the exponential operation
	    if(my_localId == 0) {
			curSpikeTime = gpu_spikeTime[spkCnt];
			curSpikeAddr = gpu_spikeAddr[spkCnt];
		}

		__syncthreads();
		
		unsigned long timeDiff = curSpikeTime-ltStamp;
		float temp = (float)(timeDiff/constNeuronParams.membraneTau);
		float decayFactor = __expf(-temp);
		
		ltStamp = curSpikeTime;

		// read the spike for spike buffer and calulate x and y addresst
		unsigned int addrx = curSpikeAddr&0xff;
		unsigned int addry = (curSpikeAddr>>8)&0xff;		

		int offSetAddrX = my_addrx - (addrx - (MAX_TEMPLATE_SIZE/2) + 1); 
		int offSetAddrY = my_addry - (addry - (MAX_TEMPLATE_SIZE/2) + 1); 
			
		// check if the neuron address is within the 
		// valid range where modification is going to happen
		// due to convolution operation
		if (offSetAddrX >= 0 &&
			offSetAddrY >= 0 &&
			offSetAddrX < MAX_TEMPLATE_SIZE &&
			offSetAddrY < MAX_TEMPLATE_SIZE ) {

				/* we can read the template and get a valid data */
				int texPos = neuronArrayId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + offSetAddrY*MAX_TEMPLATE_SIZE + offSetAddrX; 

				float weight=tex1D(template_tex, texPos); // TODO always returns 0 now

				// weights can be positive or negative based on the template type		
				refValue = refValue*decayFactor + weight; 
		}

		// neuron's membrane potential value exceeds the threshold value
		// and hence the neuron should fire and reset
		if (refValue > constNeuronParams.threshold)  {
			refValue = 0.0;
			// increment the current kernel call's firing count
			int fireId = atomicAdd(&numFiringArr[neuronArrayId], 1);		// returns the *old* value of numFiring in fireId
			
			// store the fired neuron's id in the firing table
			// TODO: include the objId along with the array for rendering by jAER
			// TODO check that templateId is correct here as MSB of addr
			if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
				firedNeuronAddr[neuronArrayId*MAX_FIRING+fireId] = (neuronArrayId<<16)+(my_addry<<8)+ my_addrx;
			}
		}
		// neuron's membrane potential value is lower than the threshold value hence saturate...
		else if (refValue < constNeuronParams.membranePotentialMin)
			refValue = constNeuronParams.membranePotentialMin;
   }

   // only one thread writes down the timeStamp value.
   // TODO: currently we store the time corresponding to the first spike
   // should this be the last spike ????
   gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx] = ltStamp;			   
   
   // write back the calculated refValue    
   gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = refValue;
}


#endif // #ifndef _TEMPLATE_KERNEL_H_
