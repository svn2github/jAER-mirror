#include "hip/hip_runtime.h"
/* jaercuda project for running AER convolution processing on the GPU. 
    From template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code. This file is #included from template.cu.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "config.h"

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

#define CHECK_COALESE(tid, addr )  if(tid%32==0) {  if((addr%256 != 0)) atomicAdd(&numErrors, 1); }

// this should not be a device pointer, it should be a host pointer which is hipMalloc'ed and which has data copied to it from the host, then bound as texture
// we cannot refer to this data here except by texFetch1d
//__device__ float *gpu_conv_template; //[MAX_NUM_OBJECT][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE]; // TODO tobi changed to hipMalloc so that memory is accessible, may cause problems
//texture <float, 1, hipReadModeElementType> template_tex;
texture <float> template_tex;
__device__ float gpu_conv_template[MAX_NUM_OBJECT][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE]; // TODO tobi changed to hipMalloc so that memory is accessible, may cause problems

__device__ float gpu_membranePotential[MAX_NUM_OBJECT][MAX_Y][MAX_X];
__device__ unsigned long gpu_lastTimeStamp[MAX_NUM_OBJECT][MAX_Y][MAX_X];
__device__ float temp_conv_value[NUM_CUDA_PACKETS][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE];

__device__ int numErrors;

__constant__ __device__  globalNeuronParams_t constNeuronParams;


__device__ int   totFiring  = 0;
__device__ int   numFiring0[MAX_NUM_OBJECT];
__device__ int   numFiring1[MAX_NUM_OBJECT];
__device__ unsigned int   firedNeuronAddr[MAX_NUM_OBJECT*MAX_FIRING]; // holds output spikes from each template

// This method implements part of the winner take-all functionality.
// The CPU calls this kernel whenever some neurons in the neuron array has fired.
// Each thread is responsible for inhibition of one neuron potential by iESynWeight amount.
// Finally each threads clips the membrane potential to stay within a specific value.
// We use a ID grid of 128x1 thread and 128 block to have simple addressing mechanism.
// 2D block of thread can also do similar computation but kernel needs slightly more address calculations.
__global__ void WTAKernel1DMO(int* numFiringAddr, int num_object)
{	
	// Thread index
	int tx = threadIdx.x;
	int bx = blockIdx.x;

	int my_addrx = tx; // this thread handles neuron (x,y)=(threadIdx.x, blockIdx.x)
	int my_addry = bx;

	__shared__ volatile int numFiring;  // shared by kernels in thread block
	
	for(int objId=0; objId < num_object; objId++) {
		
		if (tx == 0) {
			numFiring = numFiringAddr[objId]; // first thread gets the total number of spikes for this template objId and puts in shared memory
		}	

		__syncthreads(); // rest wait till here

		// only if some value exists for fired neuron
		// we update the 
		if( (volatile int) numFiring!=0 ) {	// FAQ: why cast to volatile int here? Just to be sure we are casting again to volatile. 
				 
			float temp = gpu_membranePotential[objId][my_addry][my_addrx]; // membrane potential of an LIF neuron for one template array

			temp -= constNeuronParams.iESynWeight; // reduce it by the iE weight (inhibitory to excitatory)
			
			if ( temp < constNeuronParams.membranePotentialMin )
				// clamp it to negative driving potential (negative weight can never make it fire)
				temp = constNeuronParams.membranePotentialMin; 
				
			gpu_membranePotential[objId][my_addry][my_addrx] = temp;
		}
	}	
}

// this array stores the incoming spikes from CPU...
__device__ int gpu_spikeAddr[GPU_MAX_SPIKE_PACKETS];
__device__ unsigned long gpu_spikeTime[GPU_MAX_SPIKE_PACKETS];

__device__ float debugArr[MAX_NUM_BLOCKS][100];	//used for debugging...
__device__ unsigned long debugArrInt[MAX_NUM_BLOCKS][100];	//used for debugging...

__global__ void
convNN_multiSpikeKernel( unsigned long timeStamp,	// time stamp of the first firing spike
						 int  len,					// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr,			// This memory will be reset to zero by GPU
						 int  templateId)			// value of the kernel/template that is used - TODO not used here, we're using part of blockIdx for template since all templates evaluated in parallel
					
{
	//__shared__ volatile int sh_potential;
	__shared__ volatile float sh_decayFactor;      
	//__shared__ volatile int sh_numInpSpikes;   
	// we first load the pointer... this is not useful if gpu_conv_template is statically allocated
	// if gpu_conv_template is a dynamic multi-dimensional pointer.. then it is better to do 
	// bring the pointer into register and then use integer offset to retreive corresponding data..
	// Better would be to move the gpu_conv access to texture memory...
	//float* tmp_gpu_conv_template = &gpu_conv_template[0][0][0];

	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int bx = blockIdx.x;
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.

	//We encode the object dimension in blockId itself.last 3 bit denotes
	//block number, the remaining bits denote the object number neuronArrayId
	int by = (blockIdx.y&0x7);
	int neuronArrayId = (blockIdx.y>>3);

	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = bx*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = by*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	// we get the address where we will update the number of neurons that has fired
	int* numFiring=&numFiringArr[neuronArrayId];

	// TODO only one thread needs to update this? why not use my_localId if it's unique???
	if (my_localId == 0) {		
		resetAddr[neuronArrayId]=0; // FAQ: why is the kernel doing this, what is resetAddr??? it's not used further in the kernel
									// we use a simple double buffering scheme. this address will be passed as
									// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
									// by the CPU for reseting the number of firing by this mechanism.
	}
			
   __syncthreads();

   // calculate the exponential decay value or factor into shared memory for all threads in grid to use
   // just one thread needs to evaluate it.
   if(my_localId == 0) { // don't all threads with tx==0 do this??? TODO
	   unsigned long ltStamp = gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx];
	   unsigned long timeDiff = timeStamp-ltStamp;	    
	   float temp = (float)(timeDiff/constNeuronParams.membraneTau);
	   sh_decayFactor = __expf(-temp);   
   }
   __syncthreads();

   // retreive the initial value of the membrane potential and multiply by decay value
   float refValue  = gpu_membranePotential[neuronArrayId][my_addry][my_addrx];

   refValue = refValue*sh_decayFactor;

   // read SHM_SPIKE_BUFFER_LEN from the input spike pool and
   // generate the convolution for each of the spikes
	#define SHM_SPIKE_BUFFER_LEN 128 // FAQ, why not in config.h?  this is very small number of input spikes also.
									 // we are storing the spikes in the shared memory before doing
									 // the LIF calculations. This shared memory operation can saved
									 // by either using a constant cache or a texture cache
   __shared__ volatile unsigned int sh_spike_addr[SHM_SPIKE_BUFFER_LEN]; // FAQ, why use shared memory here? why not gpu_spikeAddr?
																		 // its better to read a whole bunch of spikes to get better bandwidth performance.
																		 // it would be better to map gpu_spikeAddr to texture or constant cache.
																		 // we dont have the headache of storing spikes in shared memory
   
   int cnt=SHM_SPIKE_BUFFER_LEN; // cnt takes care of reading new set of spikes into GPU shared memory 
   for(int i=0; i < len;i++) { // for each spike from CPU

	   if(cnt==SHM_SPIKE_BUFFER_LEN) {
		   // all thread with address less then spike buffer length will fetch data from memory
		   // TODO tobi doesn't understand this scheme, what about the other threads? 
		   if(my_localId < SHM_SPIKE_BUFFER_LEN)
			   sh_spike_addr[my_localId] = gpu_spikeAddr[i+my_localId]; //params.addrV;//*(params.addr + eventId);
		   cnt=0;				
	   }
	   __syncthreads();

	   // read the spike for spike buffer and calulate x and y addresst
	   unsigned int addrx = (sh_spike_addr[cnt])&0xff;
	   unsigned int addry = (sh_spike_addr[cnt]>>8)&0xff;		
	   cnt++;

	   /* find the region of neuron array that is valid and should change */		
	   int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
	   int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
	   int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
	   int max_y = addry + (MAX_TEMPLATE_SIZE/2);

	   /* we allow negative values for min_x and min_y 
	   and should allow max_x and max_y to exceed the size,
	   but only valid neurons will be updated */

	   // TODO: remove the two conditions below.
	   if (max_x >= MAX_X )
		   max_x = MAX_X - 1;
	   if (max_y >= MAX_Y )
		   max_y = MAX_Y - 1;

	   // check if the neuron address is within the 
	   // valid range where modification is going to happen
	   // due to convolution operation
	   if (my_addrx >= min_x &&
		   my_addry >= min_y &&
		   my_addrx <= max_x &&
		   my_addry <= max_y ) {

			   // evaluate the x and y values for the template
			   int tempId_x = my_addrx - min_x;
			   int tempId_y = my_addry - min_y;

			   /* we can read the template and get a valid data */
		//	   int texPos = templateId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
			   int texPos = neuronArrayId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x; // tobi changed to neuronId which is template in this block
			   // debug
			   float weight=tex1D(template_tex, texPos); // TODO always returns 0 now
			   //float weight = tmp_gpu_conv_template[texPos];			

			   // weights can be positive or negative based on the template type		
			   refValue = refValue + weight; // tex1D (template_tex, texPos);
	   }

	   // neuron's membrane potential value exceeds the threshold value
	   // and hence the neuron should fire and reset
	   if (refValue > constNeuronParams.threshold)  {
		   refValue = 0.0;
		   // increment the current kernel call's firing count
		   int fireId = atomicAdd(numFiring, 1);		// returns the *old* value of numFiring in fireId
		   // increment the total firing count for all kernel calls until now
		   atomicAdd(&totFiring,1);						// used for debug
		   // store the fired neuron's id in the firing table
		   // TODO: include the objId along with the array for rendering by jAER
		   // TODO check that templateId is correct here as MSB of addr
		   if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
				firedNeuronAddr[neuronArrayId*MAX_FIRING+fireId] = (neuronArrayId<<16)+(my_addry<<8)+ my_addrx;
		   }
	   }
	   // neuron's membrane potential value is lower than the threshold value hence saturate...
	   else if (refValue < constNeuronParams.membranePotentialMin)
		   refValue = constNeuronParams.membranePotentialMin;
	}
	
	// only one thread writes down the timeStamp value.
	// TODO: currently we store the time corresponding to the first spike
	// should this be the last spike ????
    if(my_localId == 0)
    	gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx] = timeStamp;		

	// write back the calculated refValue 
	//if (neuronSelected)
	gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = refValue;

}


__global__ void
convNN_multiSpikeKernelNew( unsigned long timeStamp,// time stamp of the first firing spike
						 int  numInpSpikes,			// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr,			// This memory will be reset to zero by GPU
						 int  templateId)			// value of the kernel/template that is used - TODO not used here, we're using part of blockIdx for template since all templates evaluated in parallel
					
{	
	__shared__ volatile float sh_decayFactor;

	// we first load the pointer... this is not useful if gpu_conv_template is statically allocated
	// if gpu_conv_template is a dynamic multi-dimensional pointer.. then it is better to do
	// bring the pointer into register and then use integer offset to retreive corresponding data..
	// Better would be to move the gpu_conv access to texture memory...
	//float* tmp_gpu_conv_template = &gpu_conv_template[0][0][0];

	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int bx = blockIdx.x;
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.

	//We encode the object dimension in blockId itself.last 3 bit denotes
	//block number, the remaining bits denote the object number neuronArrayId
	int by = (blockIdx.y&0x7);
	int neuronArrayId = (blockIdx.y>>3);

	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = bx*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = by*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	// we get the address where we will update the number of neurons that has fired
	int* numFiring=&numFiringArr[neuronArrayId];

	// TODO only one thread needs to update this? why not use my_localId if it's unique???
	if (my_localId == 0) {		
		resetAddr[neuronArrayId]=0; // FAQ: why is the kernel doing this, what is resetAddr??? it's not used further in the kernel
									// we use a simple double buffering scheme. this address will be passed as
									// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
									// by the CPU for reseting the number of firing by this mechanism.
	}
			
   __syncthreads();

   int ltStamp = 0;

   // retreive the membrane potential
   if(my_localId == 0) {
	  ltStamp = gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx];
   }

   // retreive the initial value of the membrane potential and multiply by decay value
   float refValue  = gpu_membranePotential[neuronArrayId][my_addry][my_addrx];

   // for each spike from CPU
   for(int spkCnt=0; spkCnt < numInpSpikes;spkCnt++) {

		// only one thread within a block does the exponential operation
	    if(my_localId == 0) {
			unsigned long timeDiff = gpu_spikeTime[spkCnt]-ltStamp;
			float temp = (float)(timeDiff/constNeuronParams.membraneTau);
			sh_decayFactor = __expf(-temp);
			ltStamp = gpu_spikeTime[spkCnt];
		}

		__syncthreads();

		refValue = refValue*sh_decayFactor;

		// read the spike for spike buffer and calulate x and y addresst
		unsigned int addrx = (gpu_spikeAddr[spkCnt])&0xff;
		unsigned int addry = (gpu_spikeAddr[spkCnt]>>8)&0xff;		

		/* find the region of neuron array that is valid and should change */		
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
		int max_y = addry + (MAX_TEMPLATE_SIZE/2);

		// check if the neuron address is within the 
		// valid range where modification is going to happen
		// due to convolution operation
		if (my_addrx >= min_x &&
			my_addry >= min_y &&
			my_addrx <= max_x &&
			my_addry <= max_y ) {

				// evaluate the x and y values for the template
				int tempId_x = my_addrx - min_x;
				int tempId_y = my_addry - min_y;

				/* we can read the template and get a valid data */
				//	   int texPos = templateId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
				int texPos = neuronArrayId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x; // tobi changed to neuronId which is template in this block
				// debug
				float weight=tex1D(template_tex, texPos); // TODO always returns 0 now
				//float weight = tmp_gpu_conv_template[texPos];

				// weights can be positive or negative based on the template type		
				refValue = refValue + weight; // tex1D (template_tex, texPos);
		}

		// neuron's membrane potential value exceeds the threshold value
		// and hence the neuron should fire and reset
		if (refValue > constNeuronParams.threshold)  {
			refValue = 0.0;
			// increment the current kernel call's firing count
			int fireId = atomicAdd(numFiring, 1);		// returns the *old* value of numFiring in fireId
			// increment the total firing count for all kernel calls until now
			atomicAdd(&totFiring,1);						// used for debug
			// store the fired neuron's id in the firing table
			// TODO: include the objId along with the array for rendering by jAER
			// TODO check that templateId is correct here as MSB of addr
			if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
				firedNeuronAddr[neuronArrayId*MAX_FIRING+fireId] = (neuronArrayId<<16)+(my_addry<<8)+ my_addrx;
			}
		}
		// neuron's membrane potential value is lower than the threshold value hence saturate...
		else if (refValue < constNeuronParams.membranePotentialMin)
			refValue = constNeuronParams.membranePotentialMin;
   }

   // only one thread writes down the timeStamp value.
   // TODO: currently we store the time corresponding to the first spike
   // should this be the last spike ????
   if(my_localId == 0) {
	   gpu_lastTimeStamp[neuronArrayId][my_addry][my_addrx] = ltStamp;			   
   }

   // write back the calculated refValue    
   gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = refValue;



}

#endif // #ifndef _TEMPLATE_KERNEL_H_
