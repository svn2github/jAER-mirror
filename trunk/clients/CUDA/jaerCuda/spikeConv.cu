#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <time.h>

//#include "aer.h"

// includes, project
#include <cutil.h>

#include <Winsock2.h>
#include <Ws2tcpip.h>

#include <windows.h>

#include "config.h"

// includes, kernels
#include "spikeConv_kernel.cu"

#define CUDA   1
bool runCuda=1, tmpRunCuda=1; // tmp is for command processing

//===========================================================
// Server and client related code using Windows Socket
//===========================================================

extern char recvBuf[RECV_SOCK_BUFLEN];

//=========================================================
// Functions for jAER connections
//=========================================================

extern "C" {
	int jaerInit();
	int jaerRecv(); // fills up recvBuf with some spike data
	void jaerSendEvent(unsigned int addrx, unsigned int addry, unsigned long timeStamp, unsigned char type);
}
//==========================================================
// Functions that interface with golden reference
//==========================================================

extern "C" {
	int   templateConvInit(int selectType=TEMP_METHOD1, int templateType=TEMPLATE_DoG);
	void  computeGold( unsigned int* addr, unsigned long* timeStamp);	
	int   extractJaerRawData( unsigned int* addr, unsigned long* timeStamp, char* Data, unsigned int len);	
	void  setInitLastTimeStamp(unsigned long timeStamp, int objId=0);
}

//===========================================================
// Functions related to dumping trace info and other matlab scripts
//===========================================================
extern "C"{
	void showMembranePotential(unsigned int* spikeAddr=NULL, int spikeCnt=0);
	void printResults(FILE* fpLog);
	void dumpTemplate(FILE* fp, char* fstr);
}


//===========================================================
// Cuda related functions
//===========================================================

void  allocateDeviceMemory();
int   runjaerCUDA( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
////////////////////////////////////////////////////////////////////////////////

float inh_mem_potential[MAX_NUM_OBJECT];			// value of the inhibition potential
unsigned int  filteredSpike_addr[RECV_SOCK_BUFLEN/EVENT_LEN];	// array of filterred spike's address
unsigned long filteredSpike_timeStamp[RECV_SOCK_BUFLEN/EVENT_LEN];	// array of filtered spikes's timestamp

FILE *fpLog;						// Pointer to the log file...
long tot_fired_MO[MAX_NUM_OBJECT];  // total number of firing per neuronArray
int  inhFireCnt=0;					// total firing by inhibitory neuron
int   callCount=0;				// keeps track of number of times kernel is called
long  tot_filteredSpikes = 0;	// total number of filtered spikes since the start...
float accTimer = 0;				// total executing time is kept here...

int  num_object=MAX_NUM_OBJECT;   
// delta_time is time in us that spikes are chunked together to be sent with common timestamp. 
// increasing speeds up processing but quantizes time more.
unsigned int delta_time=1000;

int debugLevel=DEBUG_LEVEL;

void* numFiring0AddrMO;		//  points to device memory. contains list of fired neurons for odd runs
void* numFiring1AddrMO;		//	points to device memory. contains list of fired neurons for even runs
void* devPtrSpikeAddr;		//  points to device memory. cpu copies spikes to gpu through this memory
void* devPtrSpikeTime;		//  points to device memory. cpu copies spikes time to this memory in gpu

unsigned int firedNeuron_addr[MAX_SENDING_SPIKES];

// initial values for the parameters are set here...
globalNeuronParams_t hostNeuronParams={
	MEMBRANE_THRESHOLD,
	MEMBRANE_TAU,
	MEMBRANE_POTENTIAL_MIN,
	MIN_FIRING_TIME_DIFF,
	EI_SYN_WEIGHT,
	IE_SYN_WEIGHT
};

bool sendGlobalNeuronParamsEnabled=1; // flag set from control thread
bool sendTemplateEnabled=1; // set from control thread
bool stopEnabled=0;  // set by jaer command 

hipArray* cuArray; // used for texture template memory



/**********************************************************************************************************************/
/*****************************************  INITIATION  ***************************************************************/
/**********************************************************************************************************************/

/** Initialize CUDA device. If we have multiple CUDA device we are using device 0.
**/
void initiateDevice()
{
	
	int dev;
	CUT_DEVICE_INIT(argc, argv);
	CUDA_SAFE_CALL(hipSetDevice(0));
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	hipDeviceProp_t deviceProp;                              
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev)); 
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name); 
}


/** This function is to allocate device memory for the template matrix
	Also initiate the global neuron parameter structure
**/
void allocateDeviceMemory()
{	
	// allocate GPU hipArray for convolution kernel, will be bound to a texture
	// hipArray is 1d array of size sizeof(conv_template) which is M*n*n where M is # of templates and n is kernel size
	// hipArray contains float values
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL (hipMallocArray (&cuArray, &channelDesc, sizeof(conv_template), 1));
	template_tex.normalized=0; // use int lookup for texture
	template_tex.filterMode=hipFilterModePoint; // nearest texture value
	CUDA_SAFE_CALL (hipBindTextureToArray (template_tex, cuArray));

	// get device addresses for output spikes related variables
	CUDA_SAFE_CALL (hipGetSymbolAddress( &numFiring0AddrMO, HIP_SYMBOL("numFiring0")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &numFiring1AddrMO, HIP_SYMBOL("numFiring1")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &devPtrSpikeAddr, HIP_SYMBOL("gpu_spikeAddr")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &devPtrSpikeTime, HIP_SYMBOL("gpu_spikeTime")));
	
	// already initialized statically but we do it again here to be sure to get the fields
	hostNeuronParams.threshold=MEMBRANE_THRESHOLD;
	hostNeuronParams.membraneTau=MEMBRANE_TAU;
	hostNeuronParams.membranePotentialMin=MEMBRANE_POTENTIAL_MIN;
	hostNeuronParams.minFiringTimeDiff=MIN_FIRING_TIME_DIFF;
	hostNeuronParams.eISynWeight=EI_SYN_WEIGHT;
	hostNeuronParams.iESynWeight=IE_SYN_WEIGHT;
	
}


/** This function is to initialize the variables used in neural network computation
**/
void initializeNeurons()
{
	void *devPtr;
	
	// initiate the elements of the device membrane potential array
	hipDeviceSynchronize();
	CUDA_SAFE_CALL ( hipGetSymbolAddress(&devPtr, HIP_SYMBOL("gpu_membranePotential")));
#pragma warning(disable:4313)
	printf("Zeroing membrane Potentials on GPU (loc = %x, size = %d bytes)\n", devPtr, sizeof(gpu_membranePotential));
#pragma warning(default:4313)
	CUDA_SAFE_CALL( hipMemset( devPtr, 0, sizeof(membranePotential)));
	hipDeviceSynchronize();

	// initiate the elements of the device last time stamp array 
	CUDA_SAFE_CALL(hipGetSymbolAddress(&devPtr, HIP_SYMBOL("gpu_lastTimeStamp")));
#pragma warning(disable:4313)
	printf("Copying last time stamp value to GPU (loc = %x)\n", devPtr);
#pragma warning(default:4313)
	CUDA_SAFE_CALL( hipMemcpy( devPtr, &lastTimeStamp[0][0][0], sizeof(lastTimeStamp), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	
	printf("Initializing spike counter\n");
	CUDA_SAFE_CALL( hipMemset (numFiring0AddrMO, 0, sizeof(int)*MAX_NUM_OBJECT));		
	CUDA_SAFE_CALL( hipMemset (numFiring1AddrMO, 0, sizeof(int)*MAX_NUM_OBJECT));		
	hipDeviceSynchronize();
	
	CUT_CHECK_ERROR("after initializeNeurons");
	
	memset(inh_mem_potential,0,sizeof(float)*MAX_NUM_OBJECT);
	memset(tot_fired_MO,0,sizeof(long)*MAX_NUM_OBJECT);
	
	fflush(stdout);
}

/**********************************************************************************************************************/
/*****************************************  CLEAN UP  *****************************************************************/
/**********************************************************************************************************************/

/** This function cleans up the allocated device memory
**/
void cudaClean()
{
	CUDA_SAFE_CALL(hipUnbindTexture(template_tex));
	CUDA_SAFE_CALL (hipFreeArray (cuArray));
}

/**********************************************************************************************************************/
/*****************************************  PARAMETER CHANGE  *********************************************************/
/**********************************************************************************************************************/

/** This function is to online check if the parameters has been changed **/
void onlineParamChange()
{
	/** check if tmpRunCuda has been changed **/
	if(tmpRunCuda!=runCuda){
		if(tmpRunCuda){
			sendGlobalNeuronParamsEnabled=true;
			sendTemplateEnabled=true;
			runCuda=1;
		}else{
			sendGlobalNeuronParamsEnabled=false;
			sendTemplateEnabled=false;
			runCuda=0;
		}
	} // make sure we get parameters to cuda and only change runCuda here in loop

	/** check if neuron parameters has been changed **/
	if(sendGlobalNeuronParamsEnabled) {
		sendGlobalNeuronParamsEnabled=0;
		
		hipDeviceSynchronize();
		fprintf(stdout, "Copying neuron global constants to device\n");
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("constNeuronParams"),&hostNeuronParams,sizeof(globalNeuronParams_t),(size_t)0,hipMemcpyHostToDevice));
		CUT_CHECK_ERROR("Copy neuron global constants to device");
		fprintf(stderr,"Params th=%f, tau=%f, pot=%f, time=%f, eIWt=%f, iEWt=%f\n", 
			hostNeuronParams.threshold, hostNeuronParams.membraneTau, hostNeuronParams.membranePotentialMin, hostNeuronParams.minFiringTimeDiff,
			hostNeuronParams.eISynWeight, hostNeuronParams.iESynWeight);
	}
	
	/** check if template should be send to device **/
	if(sendTemplateEnabled){
		sendTemplateEnabled=0;
		templateConvInit();
		
		// send template to GPU 
		hipDeviceSynchronize();
	#pragma warning(disable:4313)
		printf("Copying templates to GPU (loc = %x, size = %d bytes)\n", cuArray, sizeof(conv_template));
	#pragma warning(default:4313)
		// copy the template to cuda texture array
		CUDA_SAFE_CALL( hipMemcpyToArray(cuArray, 0, 0, conv_template, sizeof(conv_template), hipMemcpyHostToDevice) );
		hipDeviceSynchronize();	
		
		CUT_CHECK_ERROR("after sendTemplates"); 
	}
}

/**********************************************************************************************************************/
/*****************************************  INPUT RECEIVING AND FILTERING  ********************************************/
/**********************************************************************************************************************/

/** This function is to receive events from jaer, and filter them through an refractory filter
 *  return:		-1			error occurs when REPLAY_MODE is not defined
				-2			error occurs when REPLAY_MODE is defined
				numSpikes	otherwise
 **/
int recvFilterSpikes()
{
	int iResult;
	int numEvents;
	int numSpikes;
	 
	#ifndef REPLAY_MODE
			iResult=jaerRecv(); // in recvBuf, returns immediately if input port not yet open, blocks if waiting socket open
			//jaerServerSend(recvBuf, iResult); // debug to echo back data to jaer
			if ( iResult > 0 ) {			
				numEvents=iResult/EVENT_LEN;
				if(debugLevel>0) {
					printf("Unfiltered events received: %d\n", numEvents);
					fflush(stdout);
				}
			// don't quit if we receive 0 events (tobi), just continue
			} else if ( iResult == 0 ){
				Sleep(1);
				//fprintf(stderr, "Recieved packet with 0 events, continuing loop\n");		
				return -1;
			} else {
				fprintf(stderr, "recv failed: WSAGetLastError=%d\n", WSAGetLastError());
				fflush(stderr);
				Sleep(1);
				return -1;
			}
	#else
			numEvents = 0;
			iResult = 0;
	#endif

			/** apply refractory filter to reduce number of events **/
			numSpikes = extractJaerRawData(filteredSpike_addr, filteredSpike_timeStamp, recvBuf, numEvents);		
	#ifdef REPLAY_MODE
			if (numSpikes == -1) {
				fprintf(stderr,"readNNFilter returned -1 (error), continuing\n");
				return -2;
			}
	#else
			if (numSpikes == -1) {
				fprintf(stderr,"readNNFilter returned -1 (error), continuing\n");
				fflush(stderr);
				return -1;
			}
	#endif
	
	tot_filteredSpikes += numSpikes;
	if(debugLevel>1) printf("number of spikes after refractory filter = %d\n", numSpikes);
	
	return numSpikes;
}

/**********************************************************************************************************************/
/*****************************************  OUTPUT SENDING  ***********************************************************/
/**********************************************************************************************************************/

/** This function is to send the output spikes from E and I neurons back to jaer
 *  @param:		timeStamp		current time stamp		
 *  @param:		nInhNeuronFired	number of inhibitory spikes generated within the current cycle 
 **/
void cudaCopySpikesFromGPU2jAER(unsigned long timeStamp, int* nfiredMO, char bInhNeuronFired)
{
	int net_firing = 0;
	// send the output spikes to jaer
#ifndef REPLAY_MODE
		
	// copy fired neurons	
	for ( int i=0; i < num_object; i++) {

		if(nfiredMO[i] >= MAX_FIRING) {
			if(debugLevel>-1) printf("# output spikes in kernel(%d) overflowed kernel buffer, dropping excess\n",nfiredMO[i]);
			continue;
			//TODO: copy them in smaller chunks and send to the jAER
		}

		// copy the fired array from GPU to CPU
		if (nfiredMO[i]){
			CUDA_SAFE_CALL(hipMemcpyFromSymbol(&firedNeuron_addr[0], HIP_SYMBOL("firedNeuronAddr"), sizeof(int)*nfiredMO[i], sizeof(int)*MAX_FIRING*i, hipMemcpyDeviceToHost));
			net_firing = net_firing + nfiredMO[i];
		}

		// TODO cast from raw address back to addrx, addry, shouldn't be necessary
		for(int j = 0; j < nfiredMO[i]; j++) {
			unsigned int addrx = (firedNeuron_addr[j]) & 0x7f;
			unsigned int addry = (firedNeuron_addr[j]>>8)&0x7f;
		
			// accumulate fired neuron and send to jaer
			jaerSendEvent(addrx,addry,timeStamp,i);
		}
	}

	// send one spike to jaer if any of the inhibitory neuron fires
	if(bInhNeuronFired){
		jaerSendEvent(1,1,timeStamp,0);  
		if(debugLevel > 1)
			fprintf(stdout,"cudaCopySpikesFromGPU2jAER: sent an inhibitory spike to jaer\n");
	}		
				
	if(debugLevel>1)
		fprintf(stdout,"cudaCopySpikesFromGPU2jAER: sent %d spikes to jaer\n", net_firing);
		
#else
	// ??? TODO: we bring all spike info into one firedNeuron_addr array. So all the
	// object firing is lumped into one AER display. Use different
	// color to distinguish output from different object
	int accAddr=0;
	for ( int i=0; i < num_object; i++) {

		if(accAddr >= MAX_SENDING_SPIKES) {
			printf("Total generated spikes is more than sending array size;\n");
			accAddr = 0;
		}

		accAddr += nfiredMO[i];
	}			
	//assert(accAddr == net_firing);
#endif		
}

/**********************************************************************************************************************/
/*****************************************  UPDATE INEURON  ***********************************************************/
/**********************************************************************************************************************/

/** function that updates the membrane potential of inibitory WTA neurons and send out the spike events from both E and I neurons to jaer
 * TODO: Make the inhibitory neurons also leaky. Currently the inhibitory neurons are NOT leaky...
 * @param:	numFiringAddr	the device memory address recording the number of firing per population
 * returns: b_fired_Mo_Inh:		each bit records if the global inhibitory neuron of the corresponding excitatory population has fired during the last cycle
 **/
char cudaUpdateINeuron(void* numFiringAddr, int* nfiredMO)
{
		char b_fired_Mo_Inh = 0; // a byte record which inhibitory neuron has fired
		//int n_fired_Mo_Inh = 0;	// the total number of inhibitory spikes generated within one cycle
				
		// copy the number of template layer neurons that have fired...
		CUDA_SAFE_CALL(hipMemcpy(nfiredMO, numFiringAddr, sizeof(int)*num_object, hipMemcpyDeviceToHost));
		if(debugLevel>2){
			printf("# spikes fired by object layers: ");
			for(int i=0;i<num_object;i++){
				printf("%d, ",nfiredMO[i]);
			}
			printf("\n");
		}
	
		//update inhibitory neuron membrane potentials
		for ( int i=0; i < num_object; i++) {
			tot_fired_MO[i] += nfiredMO[i]; //per object firing
			
			inh_mem_potential[i] = inh_mem_potential[i] + hostNeuronParams.eISynWeight*nfiredMO[i];
			if (inh_mem_potential[i] > hostNeuronParams.threshold) {
				inh_mem_potential[i] = 0.0;
				inhFireCnt++;	
				//n_fired_Mo_Inh++;
				b_fired_Mo_Inh = (char)(b_fired_Mo_Inh | (0x01 << i));  //set the corresponding bit to 1 if inhibitory neuron fires
			}				
		}
		
		return b_fired_Mo_Inh;
}

/**********************************************************************************************************************/
/*****************************************  GPU COMPUTING  ************************************************************/
/**********************************************************************************************************************/

// keep tracks of how many spikes that are suppied to GPU kernel
// For performance evaluation and analysis....
int paramLenArr[PARAM_LEN_SIZE];

/** GPU computation on the WTA neural network
 *  @param:		gridExcDim		grid dimension for updating excitatory membrane potentials
 *  @param:		threadExcDim	thread dimension for updating excitatory membrane potentials
 *  @param:		gridInhDim		grid dimension for updating excitatory membrane potentials after the firing of global inhibitory neuron
 *  @param:     threadInhDim	thread dimension for updating excitatory membrane potentials after the firing of global inhibitory neuron
 *	@param:		firingId		toggling between 0/1, so that one is reset (done in the kernel) and the other is used to count the number of spikes generated during one cycle 
 *  @param:		numInSpikes		the number of input spikes after refractory filtering
 **/
void GPU_MODE(dim3 gridExcDim, dim3 threadExcDim, dim3 gridInhDim, dim3 threadInhDim, int firingId, int numInSpikes){
	
	// initiate variables with the first spike
	int  index_start=0;
	int spikeLen = 1;
	unsigned long spikeTimeStampV = filteredSpike_timeStamp[0]; // set the global timestamp for packet	
	int cpu_nfiredMO[MAX_NUM_OBJECT];	// number of neurons that got fired in the last kernel call
	
	// this loop iterates over spikes in the packet, calling the kernel periodically when it has collected enough
	// spikes. after copying the spike addresses to GPU memory, it passes struct params to the kernel. 
	// then it reads the number of neurons that fired and copies back the 
	// fired neuron addresses.
	for (int spk_i = 0; spk_i < numInSpikes; spk_i++ ) {
	
		/*********************************************************/
		/****Generate input event packet and send to GPU**********/
		/*********************************************************/
		
		if (spk_i==(numInSpikes-1)){
			// this is the last spike then just go and process the bufferred spikes in the GPU.
			spikeLen++; // just increment number of spikes to be processed
		}
		else if (spikeLen == (GPU_MAX_SPIKE_PACKETS)) {
			// our buffer is full. so go and process existing spike buffer the current spike will be part of next group..
		}
		else if ((filteredSpike_timeStamp[spk_i] - spikeTimeStampV) < delta_time) {		
			// if we're not the first or last spike or at the limit, and
			// If the current time stamp of a spike is within the delta_time then
			// we buffer the spike and start reading the next spike...
			spikeLen++;
			continue;
		}				

		// Keep track of the number of spikes that are buffered and sent to CUDA. 
		// This is useful to understand the performance, as more grouping 
		// means good performance...and CUDA kernel launch overhead is reduced.
 		if(callCount < PARAM_LEN_SIZE)
			paramLenArr[callCount]=spikeLen;				
		assert(spikeLen!=0);		
		if(debugLevel>2){
			printf("copying %d spike addresses to GPU\n",spikeLen);
		}

		// copy spikes addresses to GPU
		CUDA_SAFE_CALL(hipMemcpy( devPtrSpikeAddr, &filteredSpike_addr[index_start], sizeof(int)*spikeLen, hipMemcpyHostToDevice));
		CUT_CHECK_ERROR("Copy spike addresses to GPU");
		CUDA_SAFE_CALL(hipMemcpy( devPtrSpikeTime, &filteredSpike_timeStamp[index_start], sizeof(unsigned long)*spikeLen, hipMemcpyHostToDevice));
		CUT_CHECK_ERROR("Copy spike timestamps to GPU");

#if !MEASUREMENT_MODE			
		fprintf( fpLog, "%d => len=%d t=%d a=%d\n", callCount, spikeLen, spikeTimeStampV, filteredSpike_addr[index_start]);
#endif

		/*********************************************************/
		/*******Call multi-spike kernel***************************/
		/*********************************************************/
		
		// firingId is a toggle 0/1 that is used for odd/even kernel launches.
		// the kernel writes the number of fired neurons for each template in the array
		// pointed to by numFiringArrayAddr, at the same time, it also sets the array pointed to 
		// by resetFiringArrayAddr all to zero. The host uses the numFiring values to update the WTA neurons.
		firingId = (firingId ) ? 0 : 1;
		int* numFiringArrayAddr   = (int*)((firingId)?numFiring0AddrMO:numFiring1AddrMO);
		int* resetFiringArrayAddr = (int*)((firingId)?numFiring1AddrMO:numFiring0AddrMO); // TODO, this array is unused now
		
		if(debugLevel>2){
			printf("calling multi object convNN_multiSpikeKernel with gridDim=(%d,%d,%d), threadDim=(%d,%d,%d)\n",gridExcDim.x, gridExcDim.y, gridExcDim.z, threadExcDim.x, threadExcDim.y,threadExcDim.z);
		}
		
		CUT_CHECK_ERROR("convNN_multiSpikeKernel Before kernel execution");
		convNN_multiSpikeKernelNew1 <<< gridExcDim, threadExcDim >>> (spikeLen, numFiringArrayAddr, resetFiringArrayAddr);
		CUT_CHECK_ERROR("convNN_multiSpikeKernel Kernel execution failed");	
		hipDeviceSynchronize();
		
		if(debugLevel>2) fprintf(stderr, "Kernel executed %d times...\n", callCount);
		
		//showMembranePotential(&filteredSpike_addr[index_start],spikeLen); // only for debug
		
		/***********************************************************************************/
		/********Update membrane potential of inhibitory neurons and call WTA kernel********/
		/***********************************************************************************/ 
		
		// execute updation of iNeuron potential in CPU
		// the single WTA neuron gets excited by the total number of spikes from the convolution
		char iNeuronFired = cudaUpdateINeuron(numFiringArrayAddr, cpu_nfiredMO);
		if (iNeuronFired) {
			
			if(debugLevel>2){
			printf("calling winner take all kernel WTAKernel1DMO with gridDim=(%d,%d,%d), threadDim=(%d,%d,%d)\n",gridInhDim.x, gridInhDim.y, gridInhDim.z, threadInhDim.x, threadInhDim.y,threadInhDim.z);
			}
			
			// execute iNeuronCalculations; inhibition of all other neurons in GPU
			CUT_CHECK_ERROR("WTAKernel1DMO Before kernel execution");
			WTAKernel1DMO <<< gridInhDim, threadInhDim >>> (numFiringArrayAddr, iNeuronFired);
			CUT_CHECK_ERROR("WTAKernel1DMO After kernel execution");
			hipDeviceSynchronize();
		}
		
		
		/************************* send output spikes back to jaer  ******************/
		cudaCopySpikesFromGPU2jAER(spikeTimeStampV, cpu_nfiredMO, iNeuronFired);
		
		
		/************************* update counters ***********************************/
		callCount++;
		spikeTimeStampV = filteredSpike_timeStamp[spk_i]; // store the time stamp of spike for next grouping
		spikeLen  = 1;							  // reset length
		index_start = spk_i;					  // reset the index
		
	} // iterate over spikes in this packet		
}


/**********************************************************************************************************************/
/*****************************************  MAIN FUNCTION INTERACTING WITH JAER AND CUDA GPU  *************************/
/**********************************************************************************************************************/

int
runjaerCUDA( int argc, char** argv)
{
	int setTimeStamp = 1;
	
	// this is switched alternative to 0 and 1. every time kernel is called
	int firingId = 1;
	
	// kernel evaluates multiple convolution kernels
	// setup parameters for multi-object case
	dim3 gridExcDim(8,8*num_object,1);
	dim3 threadExcDim(16,16,1);
	
	dim3 gridInhDim(128,num_object,1);
	dim3 threadInhDim(128,1,1);
	
	/** initiate jaer **/
#ifndef REPLAY_MODE
	jaerInit();
#endif

	/** initiate gpu device **/
	initiateDevice();

	/** allocates and notes down various memory in the GPU side...	 **/
	allocateDeviceMemory();
	
	// initiate timer
	unsigned int timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	
	/** Receive data until the server closes the connection **/
	do { 

		if(debugLevel>2){
			printf("*** start cycle\n");
		}
		
		/** main loop **/
		
		/** online check if the parameters has been changed **/
		onlineParamChange();
		
		/** Receive data from jaer **/
		int numSpikes = recvFilterSpikes();
		if(numSpikes == -1){
			continue;
		}else if(numSpikes == -2){
			break;
		} // if numSpikes is not smaller than 0, it implies that numSpikes > 0
		
		/** initiate the variables involved in neural network computation **/
		if(setTimeStamp == 1)	{
			//first time we need to set the timeStamp value appropriately...
			for(int i=0; i < num_object; i++) {
				setInitLastTimeStamp(*(filteredSpike_timeStamp),i);
			}
			
			if(runCuda) {				
				initializeNeurons();
			}
			
			setTimeStamp = 0;
		}

		//////////////////////////////////////////
		//        CUDA-GPU MODEL				//
		//////////////////////////////////////////
		if(runCuda) {
		
			GPU_MODE(gridExcDim,threadExcDim, gridInhDim, threadInhDim, firingId, numSpikes);
			
		} // end if(runCuda)

		//////////////////////////////////////////
		//        CPU MODEL						//
		//////////////////////////////////////////		
		else {		
			// compute reference solution
			computeGold( filteredSpike_addr, filteredSpike_timeStamp);
			showMembranePotential();
		}

	} while( stopEnabled==0 ); // until jaer tells us to exit
	
	// close timer
	CUT_SAFE_CALL( cutStopTimer(timer));
	accTimer = cutGetTimerValue(timer);
	CUT_SAFE_CALL( cutDeleteTimer( timer));
	
	if(runCuda)	
		cudaClean();
		
	printResults(fpLog);	

	fflush(stdout); // for jaer to get it
	return 0;

}

/**********************************************************************************************************************/
/*****************************************  MAIN FUNCTION  ************************************************************/
/**********************************************************************************************************************/
int main( int argc, char** argv)
{
    fpLog = fopen("sim.log","w");
	if(!fpLog) {
		fprintf(stderr, "Cannot create simulation logging file sim.log in current directory\n");		
		exit(1);
	}

	printf("starting jaercuda \n");

	/* Print out the date and time in the standard format. */
    /* Convert it to local time representation. */
    time_t curTime = time(NULL);
    struct tm *locTime;
	locTime = localtime (&curTime);     
    fprintf(fpLog, "==============================================================\n");
    fprintf(fpLog, "jAER-CUDA Simulation Log \n");
	fprintf(fpLog, "==============================================================\n");
    fputs (asctime (locTime), fpLog);
    fprintf(fpLog, "Delta time value : %d\n", delta_time);
	fflush(fpLog);    
	fflush(stdout);
	
	runjaerCUDA( argc, argv);

	fprintf(fpLog, "**END**\n");
	fclose(fpLog);

}

