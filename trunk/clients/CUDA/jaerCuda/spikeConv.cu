#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <time.h>

//#include "aer.h"

// includes, project
#include <cutil.h>

#include <Winsock2.h>
#include <Ws2tcpip.h>

#include <windows.h>

#include "config.h"

// includes, kernels
#include "spikeConv_kernel.cu"

#define CUDA   1
bool runCuda=1, tmpRunCuda=1; // tmp is for command processing

//===========================================================
// Server and client related code using Windows Socket
//===========================================================

extern char recvBuf[RECV_SOCK_BUFLEN];

//============================================
// Variables related to Audio Playback
//============================================

void waveInit();
void playAudio();
void waveClose();

//=========================================================
// Functions for jAER connections
//=========================================================

extern "C" {
	int jaerInit();
	int jaerRecv(); // fills up recvBuf with some spike data
	void jaerSendEvent(unsigned int addrx, unsigned int addry, unsigned long timeStamp, unsigned char type);
}
//==========================================================
// Functions that interface with golden reference
//==========================================================

extern "C" {
	int   templateConvInit(int selectType=TEMP_METHOD1, int templateType=TEMPLATE_DoG);
	void  computeGold( unsigned int* addr, unsigned long* timeStamp, int templateIndex);	
	int   extractJaerRawData( unsigned int* addr, unsigned long* timeStamp, char* Data, unsigned int len);	
	void  setInitLastTimeStamp(unsigned long timeStamp, int objId=0);
}

//===========================================================
// Functions related to dumping trace info and other matlab scripts
//===========================================================
extern "C"{
	void dumpResults(int objId=0);
	void showMembranePotential(unsigned int* spikeAddr=NULL, int spikeCnt=0);
	void printResults(FILE* fpLog);
	void dumpTemplate(FILE* fp, char* fstr);
}


//===========================================================
// Cuda related functions
//===========================================================

void  allocateDeviceMemory();
int   runjaerCUDA( int argc, char** argv);

extern int curTemplateIndex;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
////////////////////////////////////////////////////////////////////////////////

float inh_mem_potential[MAX_NUM_OBJECT];			// value of the inhibition potential
bool  cpu_polarity[RECV_SOCK_BUFLEN/EVENT_LEN];		// currently unused...
unsigned int  filteredSpike_addr[RECV_SOCK_BUFLEN/EVENT_LEN];	// array of filterred spike's address
unsigned long filteredSpike_timeStamp[RECV_SOCK_BUFLEN/EVENT_LEN];	// array of filtered spikes's timestamp

FILE *fpLog;						// Pointer to the log file...
long tot_fired_MO[MAX_NUM_OBJECT];  // total number of firing per neuronArray
int  inhFireCnt=0;					// total firing by inhibitory neuron
int  multi_object=MULTI_OBJECT;		
// TODO, why is num_object set here???  nothing to do with config.h or any parameter
int  num_object=MAX_NUM_OBJECT;   
// delta_time is time in us that spikes are chunked together to be sent with common timestamp. 
// increasing speeds up processing but quantizes time more.
unsigned int delta_time=1000;

int debugLevel=DEBUG_LEVEL;

void* conv_templateAddr;	//  address of convolution template kernal on device
void* numFiring0AddrMO;		//  points to device memory. contains list of fired neurons for odd runs
void* numFiring1AddrMO;		//	points to device memory. contains list of fired neurons for even runs
void* devPtrSpikeAddr;		//  points to device memory. cpu copies spikes to gpu through this memory
void* devPtrSpikeTime;		//  points to device memory. cpu copies spikes time to this memory in gpu

unsigned int firedNeuron_addr[MAX_SENDING_SPIKES];

int   callCount=0;				// keeps track of number of times kernel is called
long  tot_fired = 0;			// total fired neurons since the start...
long  tot_filteredSpikes = 0;	// total number of filtered spikes since the start...
float accTimer = 0;				// total executing time is kept here...

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    fpLog = fopen("sim.log","w");
	if(!fpLog) {
		fprintf(stderr, "Cannot create simulation logging file sim.log in current directory\n");		
		exit(1);
	}

	printf("starting jaercuda \n");

	/* Print out the date and time in the standard format. */
    /* Convert it to local time representation. */
    time_t curTime = time(NULL);
    struct tm *locTime;
	locTime = localtime (&curTime);     
    fprintf(fpLog, "==============================================================\n");
    fprintf(fpLog, "jAER-CUDA Simulation Log \n");
	fprintf(fpLog, "==============================================================\n");
    fputs (asctime (locTime), fpLog);
    fprintf(fpLog, "Delta time value : %d\n", delta_time);
	fflush(fpLog);    
	fflush(stdout);
	
	runjaerCUDA( argc, argv);

	fprintf(fpLog, "**END**\n");
	fclose(fpLog);

}

// initial values for the parameters are set here...
globalNeuronParams_t hostNeuronParams={
	MEMBRANE_THRESHOLD,
	MEMBRANE_TAU,
	MEMBRANE_POTENTIAL_MIN,
	MIN_FIRING_TIME_DIFF,
	EI_SYN_WEIGHT,
	IE_SYN_WEIGHT
};

bool sendGlobalNeuronParamsEnabled=1; // flag set from control thread
bool sendTemplateEnabled=1; // set from control thread
bool stopEnabled=0;  // set by jaer command 

hipArray* cuArray; // used for texture template memory

void allocateDeviceMemory()
{
	
//	void* devPtr;
	//int size=sizeof(float)*MAX_NUM_OBJECT*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE;
	//int size=sizeof(conv_template); // 46k bytes (48x48x5 object* 4 bytes/float), OK
	
	CUDA_SAFE_CALL(hipGetSymbolAddress(&conv_templateAddr, HIP_SYMBOL("gpu_conv_template")));     // no such address on device
	//CUDA_SAFE_CALL(hipMalloc(&conv_templateAddr, sizeof(conv_template))); // devPtr is the device (global) memory we will treat as a texture on the device
	//CUDA_SAFE_CALL(hipMemcpy(devPtr, &conv_templateAddr, sizeof(conv_templateAddr), hipMemcpyHostToDevice));

	// allocate GPU hipArray for convolution kernel, will be bound to a texture
	// hipArray is 1d array of size sizeof(conv_template) which is M*n*n where M is # of templates and n is kernel size
	// hipArray contains float values
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL (hipMallocArray (&cuArray, &channelDesc, sizeof(conv_template), 1));
	CUDA_SAFE_CALL (hipBindTextureToArray (template_tex, cuArray));
	template_tex.normalized=0; // use int lookup for texture
	template_tex.filterMode=hipFilterModePoint; // nearest texture value

   
    // CUDA_SAFE_CALL(hipMemset(conv_templateAddr,0,sizeof(conv_template)));
	// CUDA_SAFE_CALL(hipGetSymbolAddress(&template_tex,HIP_SYMBOL("template_tex"))); // template_tex is texture reference declared in device kernel
    // see http://forums.nvidia.com/index.php?showtopic=86599&hl=hipBindTexture
    // CUDA_SAFE_CALL(hipBindTexture(0, template_tex, conv_templateAddr, sizeof(conv_template))); // bind to this device memory as texture ?? where is template_tex set on host side???	
    // CUDA_SAFE_CALL(hipBindTexture(0, template_tex, conv_templateAddr)); // bind to this device memory as texture ?? where is template_tex set on host side???

	CUDA_SAFE_CALL (hipGetSymbolAddress( &numFiring0AddrMO, HIP_SYMBOL("numFiring0")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &numFiring1AddrMO, HIP_SYMBOL("numFiring1")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &devPtrSpikeAddr, HIP_SYMBOL("gpu_spikeAddr")));
	CUDA_SAFE_CALL (hipGetSymbolAddress( &devPtrSpikeTime, HIP_SYMBOL("gpu_spikeTime")));

}

#define BASE_ADDRESS_COALESE 256
#define CHECK_ADDRESS

void sendTemplates()
{
	//void *devPtr;
	hipDeviceSynchronize();
	//CUDA_SAFE_CALL( hipMemcpy( temp_conv_template,  devPtr, sizeof(conv_template), hipMemcpyDeviceToHost));
	//dumpTemplate(temp_conv_template[1], "ref_template", 1);
	//CUDA_SAFE_CALL(hipGetSymbolAddress(&devPtr, HIP_SYMBOL(gpu_conv_template)));
	CUDA_SAFE_CALL( hipMemcpy( conv_templateAddr, &conv_template[0][0][0], sizeof(conv_template), hipMemcpyHostToDevice));
	//assert(test != hipErrorInvalidSymbol);
#pragma warning(disable:4313)
	printf("Copying templates to GPU (loc = %x, size = %d bytes)\n", conv_templateAddr, sizeof(conv_template));
#pragma warning(default:4313)

	//CUDA_SAFE_CALL( hipMemcpy( conv_templateAddr, &conv_template[0][0][0], sizeof(conv_template), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpyToArray(cuArray, 0, 0, conv_template, sizeof(conv_template), hipMemcpyHostToDevice) );

	hipDeviceSynchronize();
	// do we have to bind texture again here??
	//CUDA_SAFE_CALL(hipBindTexture(0, template_tex, conv_templateAddr)); // bind to this device memory as texture ?? where is template_tex set on host side???
	sendTemplateEnabled=0;
	FILE *fp = fopen("gpu_template.m", "w");
	dumpTemplate( fp, "gpu_template.m");
	fclose(fp);
	fflush(stdout);
	CUT_CHECK_ERROR("after sendTemplates");
}

void initializeNeurons()
{
	void *devPtr;
	hipDeviceSynchronize();
	CUDA_SAFE_CALL ( hipGetSymbolAddress(&devPtr, HIP_SYMBOL("gpu_membranePotential")));
#pragma warning(disable:4313)
	printf("Zeroing membrane Potentials on GPU (loc = %x, size = %d bytes)\n", devPtr, sizeof(gpu_membranePotential));
#pragma warning(default:4313)
	CUDA_SAFE_CALL( hipMemset( devPtr, 0, sizeof(membranePotential)));
	//CUDA_SAFE_CALL( hipMemcpy( devPtr, &membranePotential[0][0][0], sizeof(membranePotential), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	CUDA_SAFE_CALL(hipGetSymbolAddress(&devPtr, HIP_SYMBOL(gpu_lastTimeStamp)));
#pragma warning(disable:4313)
	printf("Copying last time stamp value to GPU (loc = %x)\n", devPtr);
#pragma warning(default:4313)
	//CUDA_SAFE_CALL( hipMemset (devPtr, 0, sizeof(lastTimeStamp)));
	CUDA_SAFE_CALL( hipMemcpy( devPtr, &lastTimeStamp[0][0][0], sizeof(lastTimeStamp), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	
	printf("Initializing inhibitory neurons\n");
	CUDA_SAFE_CALL(hipGetSymbolAddress(&devPtr, HIP_SYMBOL(numFiring0)));	
	CUDA_SAFE_CALL( hipMemset (devPtr, 0, sizeof(int)*MAX_NUM_OBJECT));		

	CUDA_SAFE_CALL(hipGetSymbolAddress(&devPtr, HIP_SYMBOL(numFiring1)));	
	CUDA_SAFE_CALL( hipMemset (devPtr, 0, sizeof(int)*MAX_NUM_OBJECT));		

	memset(inh_mem_potential,0,sizeof(float)*MAX_NUM_OBJECT);
	memset(tot_fired_MO,0,sizeof(long)*MAX_NUM_OBJECT);

	hipDeviceSynchronize();
	
	CUT_CHECK_ERROR("after initializeNeurons");
	fflush(stdout);
}

void jaerCudaInit() 
{
	// already done in main loop //sendTemplates();
	initializeNeurons();
}

void cudaClean()
{
	
}

void cudaCopySpikesFromGPU2jAER(unsigned long timeStamp, int n_fired, int *n_firedMO)
{
		int t_ineuron_fired = 0;		
		int accAddr=0;
		
		// copy fired neurons
		if ( n_fired != 0 ) {			
			for ( int i=0; i < num_object; i++) {

				if(n_firedMO[i] >= MAX_SENDING_SPIKES) {
					if(debugLevel>-1) printf("# output spikes in kernel(%d) overflowed kernel buffer, dropping excess\n",n_firedMO[i]);
					continue;
					//TODO: copy them in smaller chunks and send to the jAER
				}

				// copy the fired array from GPU to CPU
				if (n_firedMO[i]){
					CUDA_SAFE_CALL(hipMemcpyFromSymbol(&firedNeuron_addr[0], HIP_SYMBOL("firedNeuronAddr"), sizeof(int)*n_firedMO[i], sizeof(int)*MAX_FIRING*i, hipMemcpyDeviceToHost));
				}

				// TODO cast from raw address back to addrx, addry, shouldn't be necessary
				for(int j = 0; j < n_firedMO[i]; j++) {
					unsigned int addrx = (firedNeuron_addr[j]) & 0x7f;
					unsigned int addry = (firedNeuron_addr[j]>>8)&0x7f;
				
					// accumulate fired neuron and send to jaer
					jaerSendEvent(addrx,addry,timeStamp,i);
				}

				if (t_ineuron_fired) // TODO never set???
					jaerSendEvent(1,1,timeStamp,255);  // type=1 for inhibitory neuron TODO

				accAddr += n_firedMO[i];	
			}		
		}

		if(debugLevel>1)
			fprintf(stdout,"cudaUpdateStatus: sent %d spikes to jaer\n", n_fired);

}

/*
void cudaUpdateStatus(unsigned long timeStamp, int n_fired)
{
//		int i_val = 0;
		int t_ineuron_fired = 0;
//		float f_val = 0.0;			
		
		// copy fired neurons
		if ( n_fired != 0 ) {				
				
			CUDA_SAFE_CALL (hipMemcpyFromSymbol(&firedNeuron_addr, HIP_SYMBOL("firedNeuronAddr"), sizeof(int)*n_fired, 0, hipMemcpyDeviceToHost));				
	
			for(int i = 0; i < n_fired; i++) {
			
				unsigned int addrx = (firedNeuron_addr[i]) & 0xff;
				unsigned int addry = (firedNeuron_addr[i]>>8)&0xff;
				
				// accumulate fired neuron and send to jaer
				jaerSendEvent(addrx,addry,timeStamp,0);

				if (t_ineuron_fired)
					jaerSendEvent(1,1,timeStamp,1);
			}		

		}						
}*/

int cpu_nfiredMO[MAX_NUM_OBJECT];// number of neurons that got fired in the last kernel call

// function that updates the status of inibitory WTA neurons
// TODO: Make the inhibitory neurons also leaky....
// Currently the inhibitory neurons are NOT leaky...
// returns true if any WTA neuron from any template fired, else false
bool cudaUpdateINeuron(void* numFiringAddr, unsigned long timeStamp)
{
		bool retVal = false;

		// copy the number of template layer neurons that have fired...
		CUDA_SAFE_CALL(hipMemcpy(cpu_nfiredMO, numFiringAddr, sizeof(int)*num_object, hipMemcpyDeviceToHost));
		
		int net_firing=0;
		//logic implementing inhibitory neuron for each neuronArray
		for ( int i=0; i < num_object; i++) {
			int n_fired = cpu_nfiredMO[i];
			inh_mem_potential[i] = inh_mem_potential[i] + hostNeuronParams.eISynWeight*n_fired;
			net_firing += n_fired;
			if (inh_mem_potential[i] > hostNeuronParams.threshold) {
				inh_mem_potential[i] = 0.0;
				inhFireCnt++;
				retVal= true;
			}				
			tot_fired += n_fired;
			tot_fired_MO[i] += n_fired; //per object firing
		}

		if(debugLevel>2){
			printf("# spikes fired by object layers: ");
			for(int i=0;i<num_object;i++){
				printf("%d, ",cpu_nfiredMO[i]);
			}
			printf("\n");
		}
		
#ifndef REPLAY_MODE
		// TODO, ifdef branches are wierd, one calls function to copy and send spikes to jear, the other copies spikes from GPU to the same array only
		if (net_firing) {
			cudaCopySpikesFromGPU2jAER(timeStamp, net_firing, cpu_nfiredMO);			
		}
#else
		// TODO: we bring all spike info into one firedNeuron_addr array. So all the
		// object firing is lumped into one AER display. Use different
		// color to distinguish output from different object
		int accAddr=0;
		for ( int i=0; i < num_object; i++) {

			if(accAddr >= MAX_SENDING_SPIKES) {
				printf("Total generated spikes is more than sending array size;\n");
				accAddr = 0;
			}

			//TODO: Some problem in retrieving the multiobject data
			if (cpu_nfiredMO[i]){
				CUDA_SAFE_CALL(hipMemcpyFromSymbol(&firedNeuron_addr[accAddr], HIP_SYMBOL("firedNeuronAddr"), sizeof(int)*cpu_nfiredMO[i], sizeof(int)*MAX_FIRING*i, hipMemcpyDeviceToHost));	
			}
			accAddr += cpu_nfiredMO[i];
		}			
		//assert(accAddr == net_firing);
#endif	
		return retVal;
}

/*
bool cudaUpdateINeuron(int firingId, unsigned long timeStamp)
{
		int n_fired = 0;
//		int i_val = 0;
		static float f_val = 0.0;
		bool retVal = false;
		
		// copy and then reset numFiring
		if ( firingId )
			hipMemcpy( &n_fired, numFiring1Addr, 4, hipMemcpyDeviceToHost);
		else
			hipMemcpy( &n_fired, numFiring0Addr, 4, hipMemcpyDeviceToHost);
	
		gnFired = n_fired;
		
		f_val = f_val + hostNeuronParams.eISynWeight*n_fired;
		
		if ( f_val > hostNeuronParams.threshold ) {
			f_val = 0.0;
			inhFireCnt++;
			retVal= true;
		}
		else	
			retVal = false;
		
		tot_fired += n_fired;							
		
#ifndef REPLAY_MODE
		if (n_fired) {
			cudaUpdateStatus(timeStamp, n_fired );			
		}
#else
		if ( n_fired != 0 )  {
				hipMemcpyFromSymbol(&firedNeuron_addr, HIP_SYMBOL("firedNeuronAddr"), sizeof(int)*n_fired, 0, hipMemcpyDeviceToHost);
				fprintf(stdout,"Dumping %d spikes num\n", n_fired);
				fflush(stdout);
		}
#endif	
		return retVal;
}
*/

// keep tracks of how many spikes that are suppied to GPU kernel
// For performance evaluation and analysis....
int paramLenArr[PARAM_LEN_SIZE];

float  cpuDebugArr[MAX_NUM_BLOCKS][100];	//used for debugging...
unsigned long cpuDebugArrInt[MAX_NUM_BLOCKS][100];	//used for debugging...

////////////////////////////////////////////////////////////////////////////////
//! Main function that interacts with jAER and CUDA GPU
////////////////////////////////////////////////////////////////////////////////
int
runjaerCUDA( int argc, char** argv)
{
	int iResult;
	int numEvents;	

	// Initialize CUDA device. If we have multiple CUDA device we are using device 0.
	int dev;
	CUT_DEVICE_INIT(argc, argv);
	CUDA_SAFE_CALL(hipSetDevice(0));
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	hipDeviceProp_t deviceProp;                              
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev)); 
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name); 

	// already initialized statically but we do it again here to be sure to get the fields
	hostNeuronParams.threshold=MEMBRANE_THRESHOLD;
	hostNeuronParams.membraneTau=MEMBRANE_TAU;
	hostNeuronParams.membranePotentialMin=MEMBRANE_POTENTIAL_MIN;
	hostNeuronParams.minFiringTimeDiff=MIN_FIRING_TIME_DIFF;
	hostNeuronParams.eISynWeight=EI_SYN_WEIGHT;
	hostNeuronParams.iESynWeight=IE_SYN_WEIGHT;

	// allocates and notes down various memory in the GPU side...	
	allocateDeviceMemory();

	//waveInit(); // TODO commented out because file not checked in yet


#ifndef REPLAY_MODE
	jaerInit();
#endif

//	int debugk=0;

	// CUDA GRID/BLOCK PARAMETERS ....	
	// setup execution parameters for single spike, single object case
	dim3 gridInhib(128,1,1);
	dim3 threadInhib(128,1,1);


	//if(runCuda) {
	//	printf("===> Grid configuration is (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
	//	printf("===> Block dimension is (%d, %d, %d)\n", threadDim.x, threadDim.y, threadDim.z);	

	//	fprintf(fpLog, "===> Grid configuration is (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
	//	fprintf(fpLog, "===> Block dimension is (%d, %d, %d)\n", threadDim.x, threadDim.y, threadDim.z);
	//}
	//fflush(stdout);
	
	multi_object=MULTI_OBJECT;
	unsigned int timer = 0;
	int setTimeStamp = 1;

	timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer));	
	
	// this is switched alternative to 0 and 1. every time kernel is called
	int firingId = 1;
	
		CUT_SAFE_CALL( cutStartTimer( timer));
	// Receive data until the server closes the connection
	do { 

		if(debugLevel>2){
			printf("*** start cycle\n");
		}
		// main loop
#ifndef REPLAY_MODE
		iResult=jaerRecv(); // in recvBuf, returns immediately if input port not yet open, blocks if waiting socket open
		//jaerServerSend(recvBuf, iResult); // debug to echo back data to jaer
		if ( iResult > 0 ) {			
			numEvents=iResult/EVENT_LEN;
			if(debugLevel>0) {
				printf("Unfiltered events received: %d\n", numEvents);
				fflush(stdout);
			}
		// don't quit if we receive 0 events (tobi), just continue
		} else if ( iResult == 0 ){
			Sleep(1);
			//fprintf(stderr, "Recieved packet with 0 events, continuing loop\n");		
			continue;
		} else {
			fprintf(stderr, "recv failed: WSAGetLastError=%d\n", WSAGetLastError());
			fflush(stderr);
			Sleep(1);
			continue;
		}
#else
		numEvents = 0;
		iResult = 0;
#endif

		if(tmpRunCuda!=runCuda){
			if(tmpRunCuda){
				sendGlobalNeuronParamsEnabled=true;
				sendTemplateEnabled=true;
				runCuda=1;
			}else{
				sendGlobalNeuronParamsEnabled=false;
				sendTemplateEnabled=false;
				runCuda=0;
			}
		} // make sure we get parameters to cuda and only change runCuda here in loop

		if(sendGlobalNeuronParamsEnabled) {
			sendGlobalNeuronParamsEnabled=0;
			fprintf(stdout, "Copying neuron global constants to device\n");
			CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("constNeuronParams"),&hostNeuronParams,sizeof(globalNeuronParams_t),(size_t)0,hipMemcpyHostToDevice));
			CUT_CHECK_ERROR("Copy neuron global constants to device");
			fprintf(stderr,"Params th=%f, tau=%f, pot=%f, time=%f, eIWt=%f, iEWt=%f\n", 
				hostNeuronParams.threshold, hostNeuronParams.membraneTau, hostNeuronParams.membranePotentialMin, hostNeuronParams.minFiringTimeDiff,
				hostNeuronParams.eISynWeight, hostNeuronParams.iESynWeight);
		}
		
		if(sendTemplateEnabled){
			templateConvInit();
			sendTemplates();
		}

		// apply refractory filter to reduce number of events
		int numSpikes = extractJaerRawData(filteredSpike_addr, filteredSpike_timeStamp, recvBuf, numEvents);		
#ifdef REPLAY_MODE
		if (numSpikes == -1) {
			fprintf(stderr,"readNNFilter returned -1 (error), continuing\n");
			break;
		}
#else
		if (numSpikes == -1) {
			fprintf(stderr,"readNNFilter returned -1 (error), continuing\n");
			fflush(stderr);
	//		break;
			continue;
		}
#endif
		iResult = numSpikes;		
		tot_filteredSpikes += numSpikes;
		if(debugLevel>1) printf("number of spikes after refractory filter = %d\n", numSpikes);
		
		// init jAERCUDA
		if(setTimeStamp == 1)	{
			//first time we need to set the timeStamp value appropriately...
			setInitLastTimeStamp(*(filteredSpike_timeStamp));
			setTimeStamp = 0;
			if(runCuda) {				
				jaerCudaInit();
			}
		}

		int spikeLen, trackObjectId;
		unsigned long spikeTimeStampV;

		//////////////////////////////////////////
		//        CUDA-GPU MODEL				//
		//////////////////////////////////////////
		if(runCuda) {
			int  index_start=0;
			spikeLen = 0;
			// this loop iterates over spikes in the packet, calling the kernel periodically when it has collected enough
			// spikes. after copying the spike addresses to GPU memory, it passes struct params to the kernel along with 
			// gnFired and firingId (??? what are these). then it reads the number of neurons that fired and copies back the 
			// fired neuron addresses.
			for (int spk_i = 0; spk_i < numSpikes; spk_i++ ) {
				if(spk_i==0) { /* first spike in packet */
					spikeTimeStampV = filteredSpike_timeStamp[spk_i]; // set the global timestamp for packet					
					spikeLen  = 1; // 1 spike so far
					index_start = 0; // start copying addresses from here when we transfer addresses
					continue;
				}
				else if (spk_i==(numSpikes-1)){
					// this is the last spike then just go 
					// and process the bufferred spikes in the GPU.
					spikeLen++; // just increment number of spikes to be processed
				}
				else if (spikeLen == (GPU_MAX_SPIKE_PACKETS)) {
					// our buffer is full. so go and process existing spike buffer
					// the current spike will be part of next group..
				}
				else if ((filteredSpike_timeStamp[spk_i] - spikeTimeStampV) < delta_time) {		
					// if we're not the first or last spike or at the limit, and
					// If the current time stamp of a spike is within the delta_time then
					// we buffer the spike and start reading the next spike...
					spikeLen++;
					continue;
				}

				// Keep track of the number of spikes that are buffered and sent to CUDA. 
				// This is useful to understand the performance, as more grouping 
				// means good performance...and CUDA kernel launch overhead is reduced.
				if(callCount < PARAM_LEN_SIZE)
					paramLenArr[callCount]=spikeLen;
				
				assert(spikeLen!=0);		

#ifdef REPLAY_MODE
				trackObjectId = 0;
#else
				trackObjectId = curTemplateIndex; // TODO ??? who sets this???
				//printf("curtemplateIndex = %d\n",curTemplateIndex);
#endif

				// copy spikes addresses to GPU
				if(debugLevel>2){
					printf("copying %d spike addresses to GPU\n",spikeLen);
				}

				CUDA_SAFE_CALL(hipMemcpy( devPtrSpikeAddr, &filteredSpike_addr[index_start], sizeof(int)*spikeLen, hipMemcpyHostToDevice));
				CUT_CHECK_ERROR("Copy spike addresses to GPU");
				CUDA_SAFE_CALL(hipMemcpy( devPtrSpikeTime, &filteredSpike_timeStamp[index_start], sizeof(unsigned long)*spikeLen, hipMemcpyHostToDevice));
				CUT_CHECK_ERROR("Copy spike timestamps to GPU");

#if !MEASUREMENT_MODE			
				fprintf( fpLog, "%d => len=%d t=%d a=%d\n", callCount, spikeLen, spikeTimeStampV, filteredSpike_addr[index_start]);
#endif

				// firingId is a toggle 0/1 that is used for odd/even kernel launches.
				// the kernel writes the number of fired neurons for each template in the array
				// pointed to by numFiringArrayAddr, at the same time, it also sets the array pointed to 
				// by resetFiringArrayAddr all to zero. The host uses the numFiring values to update the WTA neurons.
				// this double buffering is necessary why ??? TODO
				firingId = (firingId ) ? 0 : 1;
				int* numFiringArrayAddr   = (int*)((firingId)?numFiring0AddrMO:numFiring1AddrMO);
				int* resetFiringArrayAddr = (int*)((firingId)?numFiring1AddrMO:numFiring0AddrMO); // TODO, this array is unused now
				
				// kernel evaluates multiple convolution kernels
				// setup parameters for multi-object case
				int blockY = 8*num_object; // TODO explain this grid and thread blocking 
				dim3 gridDim(8,blockY,1);
				dim3 threadDim(16,16,1);
				if(debugLevel>2){
					printf("calling multi object convNN_multiSpikeKernel with gridDim=(%d,%d,%d), threadDim=(%d,%d,%d)\n",gridDim.x, gridDim.y, gridDim.z, threadDim.x, threadDim.y,threadDim.z);
				}
				CUT_CHECK_ERROR("convNN_multiSpikeKernel Before kernel execution");
				convNN_multiSpikeKernelNew <<< gridDim, threadDim >>>
					(spikeTimeStampV, spikeLen, numFiringArrayAddr, resetFiringArrayAddr, trackObjectId);
				// check if kernel execution generated an error
				CUT_CHECK_ERROR("convNN_multiSpikeKernel Kernel execution failed");
				if(debugLevel>2) fprintf(stderr, "Kernel executed %d times...\n", callCount);
				CUT_CHECK_ERROR("Copy spikes to GPU");
				hipDeviceSynchronize();
				showMembranePotential(&filteredSpike_addr[index_start],spikeLen); // only for debug
				// execute updation of iNeuron potential in CPU
				// the single WTA neuron gets excited by the total number of spikes from the convolution
				bool iNeuronFired = cudaUpdateINeuron(numFiringArrayAddr, spikeTimeStampV);
				if (iNeuronFired) {
					// execute iNeuronCalculations; inhibition of all other neurons in GPU
					WTAKernel1DMO <<< gridInhib, threadInhib >>> (numFiringArrayAddr, num_object);
				}
				callCount++;
				spikeTimeStampV = filteredSpike_timeStamp[spk_i]; // store the time stamp of spike for next grouping
				spikeLen  = 1;					  // reset length
				index_start = spk_i;					  // reset the index
			} // iterate over spikes in this packet
			hipDeviceSynchronize();	
		} // end if(runCuda)

		//////////////////////////////////////////
		//        CPU MODEL						//
		//////////////////////////////////////////		
		else {		
			// compute reference solution
		#ifdef REPLAY_MODE
			int templateIndex = 0;
		#else
			int templateIndex = 0; //curTemplateIndex;
		#endif		
			computeGold( filteredSpike_addr, filteredSpike_timeStamp,templateIndex);
			showMembranePotential();
		}


		// dump results
		dumpResults();

	} while( stopEnabled==0 ); // until jaer tells us to exit
	
		CUT_SAFE_CALL( cutStopTimer(timer));
		accTimer += cutGetTimerValue(timer);
	CUT_SAFE_CALL( cutDeleteTimer( timer));
		
	printResults(fpLog);	
	
	//waveClose(); // TODO commented out because not checked in

	if(runCuda)	
		cudaClean();

	//Release WinSock	
	WSACleanup();
	fflush(stdout); // for jaer to get it
	return 0;

}
