#include "hip/hip_runtime.h"
// includes, project
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>

#include "config.h"

int countMem = 0;

extern "C"	{
	void dumpTemplate(FILE* fp, char* fstr);
	void printResults(FILE* fpLog);
	void showMembranePotential(unsigned int* spikeAddr, int spikeCnt);
	void dumpResults(int objId);
}

// Function dumps the template parameters into a file
// pointed by fp. The file can be executed in matlab
void dumpTemplate(FILE* fp, char* fstr)
{
	if(fp==NULL)
		return;

	printf( "Dumping %d templates to %s\n", num_object, fstr);
	fflush(stdout); // for jaer to print this
	for(int i = 0; i < num_object; i++){		
		fprintf( fp, " template%d = [ ", i);
		for(int j=0; j < MAX_TEMPLATE_SIZE; j++) {
			for(int k=0; k < MAX_TEMPLATE_SIZE; k++) {
				fprintf( fp, " %f ", conv_template[i][j][k]);
			}
			fprintf(fp, "; \n");
		}
		fprintf(fp , " ];\nfigure;imagesc(template%d);\n\n",i);		
	}
	fflush(fp);	

}

void dumpTemplateArr(float templ[][MAX_TEMPLATE_SIZE], char* name, int id)
{
	char fname[25];
	int j,k;
	sprintf( fname, "%s%d.txt", name, id);
	static FILE* fp = fopen(fname,"w");
	
	fprintf( fp, " template%d = [ ", id);
 	for(j=0; j < MAX_TEMPLATE_SIZE; j++) {
		for(k=0; k < MAX_TEMPLATE_SIZE; k++) {
			fprintf( fp, " %f ", templ[j][k]);
		}
		fprintf(fp, "; \n");
	}
	fprintf(fp , " ];\n\n " );

	fclose(fp);
}


// only for debug, writes the membrane potentials to a file
void showMembranePotential(unsigned int* spikeAddr=NULL, int spikeCnt=0)
{
#if RECORD_MEMBRANE_POTENTIAL
		void* devPtr;		
		if((countMem >= RECORD_START && countMem <= RECORD_END))
		{
			if(runCuda)
				hipMemcpyFromSymbol(membranePotential, HIP_SYMBOL("gpu_membranePotential"), sizeof(membranePotential), 0, hipMemcpyDeviceToHost);

			char fname[100];
			sprintf(fname, "mem_pot%d.m", countMem);
			FILE* fpDumpPot;	
			fpDumpPot = fopen(fname, "w");

			fprintf( fpDumpPot, " memPot = [ " );		

			for(int i=0; i < MAX_Y; i++) {
				for(int j=0; j < MAX_X; j++) {
					fprintf( fpDumpPot, " %f ", membranePotential[0][i][j]);
				}
				fprintf(fpDumpPot, "; \n");
			}

			fprintf(fpDumpPot , " ]; " );
			fclose(fpDumpPot);

			if(spikeAddr != NULL) {
				char fname[100];
				sprintf(fname, "inpSpike%d.m", countMem);
				FILE* fpDumpPot;
				fpDumpPot = fopen(fname, "w");
				fprintf( fpDumpPot, " inpSpike = [ " );
				for(int j=0; j < spikeCnt; j++) {					
					fprintf( fpDumpPot, " %u ", spikeAddr[j]);
				}
				fprintf(fpDumpPot , " ]; " );
				fclose(fpDumpPot);	
			}		
		}	
		
		countMem++;			
#endif

}

float g_temp_conv_value[NUM_CUDA_PACKETS][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE];

void cudaCopyOutputs()
{
	void* devPtr;// = gpu_conv_template;
	
	CUDA_SAFE_CALL ( hipGetSymbolAddress(&devPtr, HIP_SYMBOL("temp_conv_value")));
#pragma warning(disable:4313)
	printf("Copying temporary template values from GPU (loc = %x size = %d\n", devPtr, sizeof(g_temp_conv_value));
#pragma warning(default:4313)
	CUDA_SAFE_CALL( hipMemcpy( g_temp_conv_value, devPtr, sizeof(g_temp_conv_value), hipMemcpyDeviceToHost));

	for(int id=0; id < NUM_CUDA_PACKETS; id++) {

//		char fname[25];
		int j,k;

		static FILE* fp = fopen("gpu_template.txt","w");

//#define G_TEMP_CONV_VALUE(i,j,k)  *(g_temp_conv_value + i*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + j*MAX_TEMPLATE_SIZE + k)
		
		fprintf( fp, " template%d = [ ", id);
 		for(j=0; j < MAX_TEMPLATE_SIZE; j++) {
			for(k=0; k < MAX_TEMPLATE_SIZE; k++) {
				fprintf( fp, " %f ", g_temp_conv_value[id][j][k]);
			}
			fprintf(fp, "; \n");
		}
		fprintf(fp , " ];\n\n " );

	}
}


void printResults(FILE* fpLog)
{
	if(!runCuda) {
		extern int cpu_totFiring;
		tot_fired = cpu_totFiring;
		extern int cpu_totFiringMO[MAX_NUM_OBJECT];
		printf(" Number of fired neurons is %d\n", tot_fired);	
		printf(" Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);					
		fprintf(fpLog, " Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);	
		fprintf(fpLog, " Number of fired neurons is %d\n", tot_fired);
		for(int i=0; i < num_object; i++) {
			printf(" Total firing in Array %d => %d\n", i, cpu_totFiringMO[i]);
			fprintf(fpLog, " Total firing in Array %d => %d\n", i, cpu_totFiringMO[i]);		
		}
	}
	else {
		printf("Kernel 1 called %d times\n", callCount);
		printf(" Total number of spikes computed : %d\n", tot_filteredSpikes);
		printf(" Number of fired neurons is %d\n", tot_fired);	
		printf(" Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);			
		fprintf(fpLog, "Kernel 1 called %d times\n", callCount);
		fprintf(fpLog, " Total number of spikes computed : %d\n", tot_filteredSpikes);
		fprintf(fpLog, " Number of fired neurons is %d\n", tot_fired);
		fprintf(fpLog, " Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);

		if (multi_object) {
			for(int i=0; i < num_object; i++) {
				printf(" Total firing in Array %d => %d\n", i, tot_fired_MO[i]);
				fprintf(fpLog, " Total firing in Array %d => %d\n", i, tot_fired_MO[i]);												
			}	
		}
	}
	
#if !VERSION_0_1
	int tot=0;
	int minLen=1000;
	int mini=0;
	int maxLen=-1;
	int maxi=0;
	int cnt=(callCount>PARAM_LEN_SIZE)?PARAM_LEN_SIZE:callCount;
	for(int i=0; i < cnt; i++) {
	  tot+= paramLenArr[i];
	  if (paramLenArr[i] < minLen) {minLen =paramLenArr[i];  mini=i; }
	  if (paramLenArr[i] > maxLen) {maxLen = paramLenArr[i]; maxi=i; }
	}
	if(runCuda) {
		printf(" Spike Distribution Per Kernel Call: \nmean(%f), min (i=%d, val=%d), max(i=%d,val=%d)\n", tot*1.0/cnt, mini, minLen, maxi,maxLen);
		fprintf(fpLog, " Spike Distribution Per Kernel Call: \nmean(%f), min (i=%d, val=%d), max(i=%d,val=%d)\n", tot*1.0/cnt, mini, minLen, maxi,maxLen);
	}
#endif

		if(runCuda) {
		int test_fired;
		hipMemcpyFromSymbol(&test_fired, HIP_SYMBOL("totFiring"), 4, 0, hipMemcpyDeviceToHost);
		printf( " Total Object scanned : %d\n", num_object);
		printf( " Total firing from Inhibition Neuron : %d\n", inhFireCnt);
		printf( " Total firing is equal to %d\n", test_fired);	
		printf( " Average firing is equal to %f\n", test_fired*1.0/callCount);
		printf( "\n\nAvg. GPU Processing time per spike: %f (ms)\n", accTimer/(tot_filteredSpikes));
		printf( "\n\nTotal GPU Processing time : %f (ms)\n", accTimer);
		fprintf( fpLog,  " Total Object scanned : %d\n", num_object);
		fprintf( fpLog,  " Total firing from Inhibition Neuron : %d\n", inhFireCnt);
		fprintf( fpLog,  " Total firing is equal to %d\n", test_fired);
		fprintf( fpLog,  "\n\nAvg. GPU Processing time per spike: %f (ms)\n", accTimer/(tot_filteredSpikes));
		fprintf( fpLog,  "\n\nTotal GPU Processing time : %f (ms)\n", accTimer);	
	}
	else {	
		extern int cpu_totFiring;
		extern int iNeuronFiringCnt;
		extern int iNeuronCallingCnt;    
		printf( " Total Object scanned : %d\n", num_object);
		printf("  INeuron Grouping Impact, Calls = %d, Firing = %d\n", iNeuronCallingCnt, iNeuronFiringCnt);
		printf( " Total firing is equal to %d\n", cpu_totFiring);
		printf( "\n\nCPU Processing time per spike: %f (ms)\n",  accTimer/(tot_filteredSpikes));
		printf( "\n\nTotal CPU Processing time : %f (ms)\n", accTimer);
		fprintf( fpLog,  " Total Object scanned : %d\n", num_object);
		fprintf( fpLog,  " Total firing from Inhibition Neuron : %d\n", iNeuronFiringCnt);
		fprintf( fpLog,  " Total firing is equal to %d\n", cpu_totFiring);
		fprintf( fpLog,  "\n\nCPU Processing time per spike: %f (ms)\n",  accTimer/(tot_filteredSpikes));
		fprintf( fpLog, "\n\nTotal CPU Processing time : %f (ms)\n", accTimer);	
	}

	fflush(stdout);  // so jaer gets it
}


void dumpResults(int objId)
{
#if DUMP_DEBUG

	char fname[100];
	sprintf(fname, "recv_packet%d.m", num_packets);
	FILE* fpDump;
	fpDump = fopen(fname, "w");
	sprintf(fname, "mem_pot%d.m", num_packets);
	FILE* fpDumpPot;	
	fpDumpPot = fopen(fname, "w");

	num_packets++;


	for(int i=0; i < MAX_Y; i++) {
		for(int j=0; j < MAX_X; j++) {		
			signed long long timeDiff = 0xFFFFFFFFLL&(prevTimeStamp-lastTimeStamp[objId][i][j]);
			if(lastTimeStamp[objId][i][j] != 0 ) {
				membranePotential[objId][i][j] = membranePotential[objId][i][j]*exp(-timeDiff/hostNeuronParams.membraneTau);
			}
			if ( membranePotential[objId][i][j] < hostNeuronParams.membranePotentialMin ) {
				membranePotential[objId][i][j] = hostNeuronParams.membranePotentialMin;
			}
		}
	}

	if(num_packets >= DEBUG_START) {

		fprintf( fpDumpPot, " memPot = [ " );
		for(int i=0; i < MAX_Y; i++) {
			for(int j=0; j < MAX_X; j++) {
				fprintf( fpDumpPot, " %f ", membranePotential[objId][i][j]);
			}
			fprintf(fpDumpPot, "; \n");
		}
		
		fprintf(fpDumpPot , " ]; " );

		/*fprintf( fpDumpPot, " excSyn = [ " );
		for(int i=0; i < MAX_Y; i++) {
			for(int j=0; j < MAX_X; j++) {
				fprintf( fpDumpPot, " %f ", excSyn[i][j]);
			}
			fprintf(fpDumpPot, "; \n");
		}
		
		fprintf(fpDumpPot , " ]; " );

		fprintf( fpDumpPot, " inhSyn = [ " );
		for(int i=0; i < MAX_Y; i++) {
			for(int j=0; j < MAX_X; j++) {
				fprintf( fpDumpPot, " %f ", inhSyn[i][j]);
			}
			fprintf(fpDumpPot, "; \n");
		}
		
		fprintf(fpDumpPot , " ]; " );*/

		fflush(fpDumpPot);
		fclose(fpDumpPot);

	}

	if(num_packets > DEBUG_END) {
		CUT_EXIT(argc, argv);
	}

#endif
}