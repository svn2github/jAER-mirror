#include "hip/hip_runtime.h"
/* jaercuda project for running AER convolution processing on the GPU. 
    From template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code. This file is #included from template.cu.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "config.h"


// this should not be a device pointer, it should be a host pointer which is hipMalloc'ed and which has data copied to it from the host, then bound as texture
// we cannot refer to this data here except by texFetch1d
texture <float> template_tex;
//__device__ float gpu_conv_template[MAX_NUM_TEMPLATE][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE]; // TODO tobi changed to hipMalloc so that memory is accessible, may cause problems

__device__ float gpu_membranePotential[MAX_NUM_TEMPLATE][MAX_Y][MAX_X];
__device__ unsigned long gpu_lastTimeStamp[MAX_Y][MAX_X];

__constant__ __device__ globalNeuronParams_t constNeuronParams;

__constant__ __device__ int const_num_object; 
__constant__ __device__ int const_radius_loc_inh;
__constant__ __device__ int const_size_loc_inh;

__device__ int   numFiring0[MAX_NUM_TEMPLATE];
__device__ int   numFiring1[MAX_NUM_TEMPLATE];     
__device__ int   gpu_curNumFiring[MAX_NUM_TEMPLATE][MAX_Y][MAX_X]; // the number of generated spikes during current kernel cycle
__device__ unsigned int   firedNeuronAddr[MAX_NUM_TEMPLATE*MAX_FIRING]; // holds output spikes from each template

/** This method implements part of the winner-take-all functionality within each population.
 * The CPU calls this kernel whenever some neurons in the neuron array has fired.
 * Each thread is responsible for inhibition of one neuron potential by iESynWeight amount.
 * Finally each thread clamps the membrane potential to stay within a specific value.
 * We use a ID grid of 128x1 thread and 128*num_object block to have simple addressing mechanism.
 * 2D block of thread can also do similar computation but kernel needs slightly more address calculations.
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	iNeuronFired		each bit records if the global inhibitory neuron of the corresponding excitatory population has fired during the last cycle
 **/
__global__ void WTAKernelMO(int* numFiringAddr, char iNeuronFired) 
{	
	// Thread index
	int my_addrx = threadIdx.x; // this thread handles neuron (x,y)=(threadIdx.x, blockIdx.x)
	int my_addry = blockIdx.x;
	int neuronArrayId = blockIdx.y;
	
	// check if the global inhibitory neuron of a particular population fired
	char iFired = (char)(iNeuronFired & (0x01<<neuronArrayId));

	if(iFired){
		float temp = gpu_membranePotential[neuronArrayId][my_addry][my_addrx]; // membrane potential of an LIF neuron for one template array

		temp -= constNeuronParams.iESynWeight; // reduce it by the iE weight (inhibitory to excitatory)
		
		if ( temp < constNeuronParams.membranePotentialMin )
			// clamp it to negative driving potential (negative weight can never make it fire)
			temp = constNeuronParams.membranePotentialMin; 
			
		gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = temp;
	}		
}

/** This method implements part of the global winner-take-all functionality among populations.
 * The CPU calls this kernel whenever some neurons in the neuron array has fired.
 * Each thread is responsible for inhibition of one neuron potential by iESynWeight amount.
 * Finally each thread clamps the membrane potential to stay within a specific value.
 * We use a ID grid of 128x1 thread and 128*num_object block to have simple addressing mechanism.
 * 2D block of thread can also do similar computation but kernel needs slightly more address calculations.
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	n_iNeuronFired		the number of spikes the global inhibitory neuron fired during the current cycle
 **/
__global__ void WTAKernelMOGlob(int* numFiringAddr, int n_iNeuronFired) 
{	
	// Thread index
	int my_addrx = threadIdx.x; // this thread handles neuron (x,y)=(threadIdx.x, blockIdx.x)
	int my_addry = blockIdx.x;
	int neuronArrayId = blockIdx.y;
	
	// check if the global inhibitory neuron of a particular population fired

	if(n_iNeuronFired != 0){
		float temp = gpu_membranePotential[neuronArrayId][my_addry][my_addrx]; // membrane potential of an LIF neuron for one template array

		temp -= n_iNeuronFired*constNeuronParams.iESynWeight; // reduce it by the iE weight (inhibitory to excitatory)
		
		if ( temp < constNeuronParams.membranePotentialMin )
			// clamp it to negative driving potential (negative weight can never make it fire)
			temp = constNeuronParams.membranePotentialMin; 
			
		gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = temp;
	}		
}

// this array stores the incoming spikes from CPU...
__device__ int gpu_spikeAddr[GPU_MAX_SPIKE_PACKETS];
__device__ unsigned long gpu_spikeTime[GPU_MAX_SPIKE_PACKETS];


/** This kernel is to update the excitatory neurons within each population
 * @param:	numInpSpikes		total number of input spikes within current cycle
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	resetAddr			the array recording the number of spikes generated within each population during the last cycle, needs to be reset during the kernel call
 **/
__global__ void
convNN_multiSpike_Kernel(int  numInpSpikes,			// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr)			// This memory will be reset to zero by GPU						
					
{	
	
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = blockIdx.x*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = (blockIdx.y&0x7)*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
	
	//We encode the object dimension in blockId itself.last 3 bit denotes
	//block number, the remaining bits denote the object number neuronArrayId
	int neuronArrayId = (blockIdx.y>>3);
	
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	// only one thread is updated to reduce the global memory access
	if (my_localId == 0) {		
		resetAddr[neuronArrayId]=0; // FAQ: why is the kernel doing this, what is resetAddr??? it's not used further in the kernel
									// we use a simple double buffering scheme. this address will be passed as
									// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
									// by the CPU for reseting the number of firing by this mechanism.
	}
		
   __syncthreads();
   
   __shared__ unsigned long curSpikeTime;
   __shared__ unsigned int curSpikeAddr;

   unsigned long ltStamp = 0;
   // retreive the membrane potential
   ltStamp = gpu_lastTimeStamp[my_addry][my_addrx];
   
   // retreive the initial value of the membrane potential and multiply by decay value
   float refValue  = gpu_membranePotential[neuronArrayId][my_addry][my_addrx];

   // for each spike from CPU
   for(int spkCnt=0; spkCnt < numInpSpikes;spkCnt++) {

		// only one thread within a block does the exponential operation
	    if(my_localId == 0) {
			curSpikeTime = gpu_spikeTime[spkCnt];
			curSpikeAddr = gpu_spikeAddr[spkCnt];
		}

		__syncthreads();
		
		unsigned long timeDiff = curSpikeTime-ltStamp;
		float temp = (float)(timeDiff/constNeuronParams.membraneTau);
		float decayFactor = __expf(-temp);
	
		// read the spike for spike buffer and calulate x and y address
		unsigned int addrx = curSpikeAddr&0xff;
		unsigned int addry = (curSpikeAddr>>8)&0xff;		

		int offSetAddrX = my_addrx - (addrx - (MAX_TEMPLATE_SIZE/2) + 1); 
		int offSetAddrY = my_addry - (addry - (MAX_TEMPLATE_SIZE/2) + 1); 
			
		// check if the neuron address is within the 
		// valid range where modification is going to happen
		// due to convolution operation
		if (offSetAddrX >= 0 &&
			offSetAddrY >= 0 &&
			offSetAddrX < MAX_TEMPLATE_SIZE &&
			offSetAddrY < MAX_TEMPLATE_SIZE ) {

			/* we can read the template and get a valid data */
			int texPos = neuronArrayId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + offSetAddrY*MAX_TEMPLATE_SIZE + offSetAddrX; 

			float weight=tex1D(template_tex, texPos); // TODO always returns 0 now

			// weights can be positive or negative based on the template type		
			refValue = refValue*decayFactor + weight; 
		
			ltStamp = curSpikeTime;

			// neuron's membrane potential value exceeds the threshold value
			// and hence the neuron should fire and reset
			if (refValue > constNeuronParams.threshold)  {
				refValue = 0.0;
				// increment the current kernel call's firing count
				int fireId = atomicAdd(&numFiringArr[neuronArrayId], 1);		// returns the *old* value of numFiring in fireId
				
				// store the fired neuron's id in the firing table
				// TODO: include the objId along with the array for rendering by jAER
				// TODO check that templateId is correct here as MSB of addr
				if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
					firedNeuronAddr[neuronArrayId*MAX_FIRING+fireId] = (neuronArrayId<<16)+(my_addry<<8)+ my_addrx;
				}
			}
			// neuron's membrane potential value is lower than the threshold value hence saturate...
			else if (refValue < constNeuronParams.membranePotentialMin)
				refValue = constNeuronParams.membranePotentialMin;
		}
   }

   // only one thread writes down the timeStamp value.
   // TODO: currently we store the time corresponding to the first spike
   // should this be the last spike ????
   if(neuronArrayId == 0)
		gpu_lastTimeStamp[my_addry][my_addrx] = ltStamp;			   
   
   // write back the calculated refValue    
   gpu_membranePotential[neuronArrayId][my_addry][my_addrx] = refValue;
}

/** This kernel is to update neurons (from different populations) at the same position together, and add in lateral inhibition between these neurons for each input spike
 * @param:	numInpSpikes		total number of input spikes within current cycle
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	resetAddr			the array recording the number of spikes generated within each population during the last cycle, needs to be reset during the kernel call
 **/
__global__ void
convNN_LocalWTA_Kernel(int  numInpSpikes,		// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr)			// This memory will be reset to zero by GPU						
					
{	
	
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = blockIdx.x*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = (blockIdx.y&0x7)*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
		
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	int i,j,spkCnt;
	// only one thread is updated to reduce the global memory access
	if (my_localId == 0) {
		for(i = 0 ; i < const_num_object; i++){
			resetAddr[i]=0; // FAQ: why is the kernel doing this, what is resetAddr??? it's not used further in the kernel
										// we use a simple double buffering scheme. this address will be passed as
										// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
										// by the CPU for reseting the number of firing by this mechanism.
		}
	}
		
   __syncthreads();
   
   __shared__ unsigned long curSpikeTime;
   __shared__ unsigned int curSpikeAddr;

   unsigned long ltStamp = 0;
   // retreive the membrane potential
   ltStamp = gpu_lastTimeStamp[my_addry][my_addrx];
   
   // retreive the initial value of the membrane potential and multiply by decay value
	float refValue[MAX_NUM_TEMPLATE];
	char b_NeuronFired; // each bit record if the neuron in each population is fired due to the current input spike
	for(i = 0; i < const_num_object; i++){
		refValue[i]  = gpu_membranePotential[i][my_addry][my_addrx];
	}
	
	// calulate the membrane potential for each input spikes
    for(spkCnt=0; spkCnt < numInpSpikes;spkCnt++) {
   		
   		if(my_localId == 0) {
			curSpikeTime = gpu_spikeTime[spkCnt];
			curSpikeAddr = gpu_spikeAddr[spkCnt];
		}

		__syncthreads();
		
		b_NeuronFired = 0; // reset the spike flags
		
		unsigned long timeDiff = curSpikeTime-ltStamp;
		float decayFactor = __expf((float)(timeDiff/constNeuronParams.membraneTau)*(-1.0f));
		   								
		// read the spike for spike buffer and calulate x and y addresst
		unsigned int addrx = curSpikeAddr&0xff;
		unsigned int addry = (curSpikeAddr>>8)&0xff;		

		int offSetAddrX = my_addrx - (addrx - (MAX_TEMPLATE_SIZE/2) + 1); 
		int offSetAddrY = my_addry - (addry - (MAX_TEMPLATE_SIZE/2) + 1); 
			
		// check if the neuron address is within the 
		// valid range where modification is going to happen
		// due to convolution operation
		if (offSetAddrX >= 0 &&
			offSetAddrY >= 0 &&
			offSetAddrX < MAX_TEMPLATE_SIZE &&
			offSetAddrY < MAX_TEMPLATE_SIZE ) {
	
			ltStamp = curSpikeTime;
			
			for(i = 0 ; i < const_num_object; i++){
				/* we can read the template and get the weight */
				int texPos = i*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + offSetAddrY*MAX_TEMPLATE_SIZE + offSetAddrX; 
				float weight=tex1D(template_tex, texPos); // TODO always returns 0 now
				
				// weights can be positive or negative based on the template type		
				refValue[i] = refValue[i]*decayFactor + weight; 
			
				// neuron's membrane potential value exceeds the threshold value
				// and hence the neuron should fire and reset
				if (refValue[i] > constNeuronParams.threshold)  {
					refValue[i] = 0.0;
					// increment the current kernel call's firing count
					int fireId = atomicAdd(&numFiringArr[i], 1);		// returns the *old* value of numFiring in fireId
					
					b_NeuronFired = (char)(b_NeuronFired | (0x01 << i));
					
					// store the fired neuron's id in the firing table
					// TODO: include the objId along with the array for rendering by jAER
					// TODO check that templateId is correct here as MSB of addr
					if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
						firedNeuronAddr[i*MAX_FIRING+fireId] = (i<<16)+(my_addry<<8)+ my_addrx;
					}
				}
				// neuron's membrane potential value is lower than the threshold value hence saturate...
				else if (refValue[i] < constNeuronParams.membranePotentialMin)
					refValue[i] = constNeuronParams.membranePotentialMin;
			}
			
			// if a spike is generated, inhibit all the neurons at the same location but in other populations 
			if(b_NeuronFired != 0){	 
				for(i = 0; i < const_num_object; i++){
					char neuronFired = (b_NeuronFired >> i) & (0x01);
					for(j = 0; j < const_num_object; j++){
						if(i != j){
							refValue[j] = refValue[j] - neuronFired * constNeuronParams.iESynWeight;
							if (refValue[i] < constNeuronParams.membranePotentialMin)
								refValue[i] = constNeuronParams.membranePotentialMin;
						}
					}
				}
			}
		}
	}
	
	for(i = 0; i < const_num_object; i++){   
	   // write back the calculated refValue    
	   gpu_membranePotential[i][my_addry][my_addrx] = refValue[i];
	}

   // only one thread writes down the timeStamp value.
   gpu_lastTimeStamp[my_addry][my_addrx] = ltStamp;			   
   
   
}

/** This kernel is to update neurons (from different populations) at the same position together, and add in lateral inhibition between these neurons for each input spike, 
 *  local inhibition between populations for each kernel call
 * @param:	numInpSpikes		total number of input spikes within current cycle
 * @param:  numFiringAddr		the array recording the number of spikes generated within each population during the current cycle
 * @param:	resetAddr			the array recording the number of spikes generated within each population during the last cycle, needs to be reset during the kernel call
 **/
__global__ void
convNN_LocalWTA_Kernel1(int  numInpSpikes,		// length of the spikes given to GPU
						 int* numFiringArr,			// pointer to number of fired neurons
													// initial value is zero before calling
						 int* resetAddr)			// This memory will be reset to zero by GPU						
					
{	
	
	//TODO: This code is specific to an image of size 128, with 8 blocks
	//each operating 16x16 pixel array.
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = blockIdx.x*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = (blockIdx.y&0x7)*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;
		
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block

	int i,j,k,spkCnt;
	
	// only one thread is updated to reduce the global memory access
	if (my_localId == 0) {
		for(i = 0 ; i < const_num_object; i++){
			resetAddr[i]=0; 			// we use a simple double buffering scheme. this address will be passed as
										// numFiringArr address during the next kernel call. We can save a hipMemcpy or hipMemset
										// by the CPU for reseting the number of firing by this mechanism.
		}
	}

   __syncthreads();
	   
   __shared__ unsigned long curSpikeTime;
   __shared__ unsigned int curSpikeAddr;

   unsigned long ltStamp = 0;
   // retreive the membrane potential
   ltStamp = gpu_lastTimeStamp[my_addry][my_addrx];
   
   // retreive the initial value of the membrane potential and multiply by decay value
	float refValue[MAX_NUM_TEMPLATE];
	int curNumFiring[MAX_NUM_TEMPLATE]; //there are two functions for this variable
										// 1. accumulate the number of spikes generated during last kernel call from other populations within a local area centered by the neuron's location
										//	  this is to calculate the total amount of inhibition from last kernel call
										// 2. accumulate the number of spikes generated during the current kernel call from each population, and before quitting the kernel, write it back to gpu_curNumFiring
	char b_NeuronFired; // each bit record if the neuron in each population is fired due to the current input spike
	
	// count the number of inhibitory input spikes from last kernel cycle
	for(i = 0; i < const_num_object; i++){
		refValue[i]  = gpu_membranePotential[i][my_addry][my_addrx]; // get the membrane potential
		curNumFiring[i] = 0; // reset the counter for the accumulation of spikes generated during last kernel call from other populations
		
		for(k = -const_radius_loc_inh; k <= const_radius_loc_inh; k++){ // check the local area centered by the neuron's location
			int tmp_addrx = my_addrx + k;
			int tmp_addry = my_addry + k;
			if(tmp_addrx >= 0
				& tmp_addry >= 0
				& tmp_addrx < MAX_X
				& tmp_addry < MAX_Y){	// boundary check
				
				for(j = 0; j < const_num_object; j++){	// accumulate all the spikes generated from other populations
					if(j != i){
						curNumFiring[i] += gpu_curNumFiring[j][tmp_addry][tmp_addrx];
					}
				}
			}
		}
	}
	
	// calulate the membrane potential for each input spikes
    for(spkCnt=0; spkCnt < numInpSpikes;spkCnt++) {
   		
   		if(my_localId == 0) {
			curSpikeTime = gpu_spikeTime[spkCnt];
			curSpikeAddr = gpu_spikeAddr[spkCnt];
		}

		__syncthreads();
		
		b_NeuronFired = 0; // reset the spike flags
		
		unsigned long timeDiff = curSpikeTime-ltStamp;
		float decayFactor = __expf((float)(timeDiff/constNeuronParams.membraneTau)*(-1.0f));
		
		// at the beginning of the kernel call, calculate the amount of inhibition from the last kernel call
		if(spkCnt == 0){
			for(i = 0; i < const_num_object; i++){
				refValue[i] = (refValue[i] - curNumFiring[i] * constNeuronParams.iESynWeight) * decayFactor; // do not check the lower bound here, to maintain the real efficacy of inhibition
				curNumFiring[i] = 0;	// reset the counter again to be used for spike counting during current kernel call
			}
			
			// reset decay factor and last time stamp since the decay has been calculated
			decayFactor = 1;	
			ltStamp = curSpikeTime;	
		}
		   								
		// read the spike for spike buffer and calulate x and y addresst
		unsigned int addrx = curSpikeAddr&0xff;
		unsigned int addry = (curSpikeAddr>>8)&0xff;		

		int offSetAddrX = my_addrx - (addrx - (MAX_TEMPLATE_SIZE/2) + 1); 
		int offSetAddrY = my_addry - (addry - (MAX_TEMPLATE_SIZE/2) + 1); 
			
		// check if the neuron address is within the 
		// valid range where modification is going to happen
		// due to convolution operation
		if (offSetAddrX >= 0 &&
			offSetAddrY >= 0 &&
			offSetAddrX < MAX_TEMPLATE_SIZE &&
			offSetAddrY < MAX_TEMPLATE_SIZE ) {
	
			ltStamp = curSpikeTime;
			
			for(i = 0 ; i < const_num_object; i++){
				/* we can read the template and get the weight */
				int texPos = i*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + offSetAddrY*MAX_TEMPLATE_SIZE + offSetAddrX; 
				float weight=tex1D(template_tex, texPos); // TODO always returns 0 now
				
				// weights can be positive or negative based on the template type		
				refValue[i] = refValue[i]*decayFactor + weight; 
			
				// neuron's membrane potential value exceeds the threshold value
				// and hence the neuron should fire and reset
				if (refValue[i] > constNeuronParams.threshold)  {
					refValue[i] = 0.0;
					// increment the current kernel call's firing count
					int fireId = atomicAdd(&numFiringArr[i], 1);		// returns the *old* value of numFiring in fireId
					
					b_NeuronFired = (char)(b_NeuronFired | (0x01 << i)); // set the corresponding bit
					curNumFiring[i]++;	// increase the spike counter by 1
					
					// store the fired neuron's id in the firing table
					// TODO: include the objId along with the array for rendering by jAER
					// TODO check that templateId is correct here as MSB of addr
					if(fireId<MAX_FIRING){ //  bounds check on output, TODO check is that correct with multi templates???
						firedNeuronAddr[i*MAX_FIRING+fireId] = (i<<16)+(my_addry<<8)+ my_addrx;
					}
				}
				// neuron's membrane potential value is lower than the threshold value hence saturate...
				else if (refValue[i] < constNeuronParams.membranePotentialMin)
					refValue[i] = constNeuronParams.membranePotentialMin;
			}
			
			// if a spike is generated, inhibit all the neurons at the same location but in other populations 
			if(b_NeuronFired != 0){	
				for(i = 0; i < const_num_object; i++){
					char neuronFired = (b_NeuronFired >> i) & (0x01);
					for(j = 0; j < const_num_object; j++){
						if(i != j){
							refValue[j] = refValue[j] - neuronFired * constNeuronParams.iESynWeight;
							if (refValue[i] < constNeuronParams.membranePotentialMin)
								refValue[i] = constNeuronParams.membranePotentialMin;
						}
					}
				}
			}
		}
	}
	
	for(i = 0; i < const_num_object; i++){   
	   // write back the calculated refValue    
	   gpu_membranePotential[i][my_addry][my_addrx] = refValue[i];
	   gpu_curNumFiring[i][my_addry][my_addrx] = curNumFiring[i];
	}

   // only one thread writes down the timeStamp value.
   gpu_lastTimeStamp[my_addry][my_addrx] = ltStamp;			   
   
   
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
