#include "hip/hip_runtime.h"
/* jaercuda project for running AER convolution processing on the GPU. 
    From template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code. This file is #included from template.cu.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "config.h"

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

#define CHECK_COALESE(tid, addr )  if(tid%32==0) {  if((addr%256 != 0)) atomicAdd(&numErrors, 1); }

__device__ float *gpu_conv_template; //[MAX_NUM_OBJECT][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE];
texture <float> template_tex;

__device__ float gpu_membranePotential[MAX_NUM_OBJECT][MAX_Y][MAX_X];
__device__ unsigned long gpu_lastTimeStamp[MAX_NUM_OBJECT][MAX_Y][MAX_X];
//__device__ const float gpu_objSizeArray[MAX_NUM_OBJECT] = {16.0,8.0};	
__device__ const float gpu_objSizeArray[] = {15.0,7.0,20.0,19.0,18.0,12.0,11.0,10.0};	//ball size
__device__ float temp_conv_value[NUM_CUDA_PACKETS][MAX_TEMPLATE_SIZE][MAX_TEMPLATE_SIZE];

__device__ int numErrors;

__constant__ globalNeuronParams_t constNeuronParams;

#define MAX_FIRING 1000

__device__ int   totFiring  = 0;
__device__ int   numFiring0 = 0;
__device__ int   numFiring1 = 0;
__device__ int   numFiring2 = 0;
__device__ int   totFiringMO[MAX_NUM_OBJECT];
__device__ int   numFiring0MO[MAX_NUM_OBJECT];
__device__ int   numFiring1MO[MAX_NUM_OBJECT];
__device__ int   numFiring2MO[MAX_NUM_OBJECT];
__device__ unsigned int   firedNeuronAddr[MAX_NUM_OBJECT*MAX_FIRING];
__device__ float gpu_iNeuronPotential[MAX_NUM_OBJECT];
__device__ int   gpu_iNeuronFired;

__global__ void WTAKernel1D()
{
	// TODO: current scheme works with only
	// one object map
	int objId  = 0;
	
	// Thread index
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	
	int my_addrx = tx;
	int my_addry = bx;
			 
	float temp = gpu_membranePotential[objId][my_addry][my_addrx];

	temp -= constNeuronParams.iESynWeight;
		
	if ( temp < constNeuronParams.membranePotentialMin )
		temp = constNeuronParams.membranePotentialMin;
			
	gpu_membranePotential[objId][my_addry][my_addrx] = temp;

}	

__global__ void WTAKernel1DMO(int firingId, int num_object)
{	
	// Thread index
	int tx = threadIdx.x;
	int bx = blockIdx.x;

	int my_addrx = tx;
	int my_addry = bx;
	__shared__ volatile int numFiring;
	
	for(int objId=0; objId < num_object; objId++) {
		
		if (tx == 0) {
			if (firingId ) 
				numFiring = numFiring1MO[objId];
			else
				numFiring = numFiring0MO[objId];
		}
		
		__syncthreads();
		
		if( (volatile int) numFiring!=0 ) {			
				 
			float temp = gpu_membranePotential[objId][my_addry][my_addrx];

			temp -= constNeuronParams.iESynWeight;
			
			if ( temp < constNeuronParams.membranePotentialMin )
				temp = constNeuronParams.membranePotentialMin;
				
			gpu_membranePotential[objId][my_addry][my_addrx] = temp;
		}
	}
}	
__global__ void WTAKernel()
{
	int objId = 0;  // TODO: currently we support inhibition only in 1st membranePotential Array
	
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int my_addrx = bx*blockDim.x + tx;
	int my_addry = by*blockDim.y + ty;	
	  
	float temp_iNeuronPotential = gpu_iNeuronPotential[objId] + constNeuronParams.eISynWeight*numFiring0;

	if (temp_iNeuronPotential > constNeuronParams.threshold) {
		//CHECK_COALESE( (blockDim.x*ty+ tx), (int)&gpu_membranePotential[objId][my_addry][my_addrx]);
		gpu_membranePotential[objId][my_addry][my_addrx] -= constNeuronParams.iESynWeight;
			
		if(gpu_membranePotential[objId][my_addry][my_addrx] < constNeuronParams.membranePotentialMin ) 
			gpu_membranePotential[objId][my_addry][my_addrx] = constNeuronParams.membranePotentialMin;		
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
generateGlobalTemplateKernel( cudaParameters_t params, int prevFired, int firingId)
{
	int objId = 0;
	
    // Block index
    int bx = blockIdx.x;//%3;
    int by = blockIdx.y;///3;
	int bz = blockIdx.z;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	// location of conv. blocks
//	int eventId = bz;
	int subTemplateId_x = bx;
	int subTemplateId_y = by;
	
	int* numFiring;
	
	if ( firingId )
		numFiring = &numFiring1;
	else
		numFiring = &numFiring0;

	if ( prevFired ) {
		if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.y == 0)) {
			if (firingId)
				numFiring0 = 0;
			else
				numFiring1 = 0;
		}
	}	
	
	if ( bz < params.len )  {

		unsigned int addr  = params.addrV;//*(params.addr + eventId);
		unsigned int addrx = (addr)&0xff;
		unsigned int addry = (addr>>8)&0xff;
		unsigned long timeStamp  = params.timeStampV ; //*(params.timeStamp + eventId);
		
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		
		int my_addrx = min_x + subTemplateId_x*MAX_SUB_TEMPLATE_SIZE_X + tx; 
		int my_addry = min_y + subTemplateId_y*MAX_SUB_TEMPLATE_SIZE_Y + ty;
				
 		if  ( (my_addrx > 0) &&
			  (my_addrx < MAX_X) &&
			  (my_addry > 0) &&
			  (my_addry < MAX_Y))  {
		
			unsigned long ltStamp = gpu_lastTimeStamp[objId][my_addry][my_addrx];
		
			// we are within logical boundary of frame
			//signed long long timeDiff = 0xFFFFFFFFLL&(timeStamp-ltStamp); 
		    unsigned long timeDiff = timeStamp-ltStamp;
		    //unsigned long tDiff = timeStamp - ltStamp;	
			//if ( tDiff & 0x80000000L)
			//	tDiff = 0;
			
			//signed long timeDiff = (signed long)(-tDiff);
		    
			float temp = (float)(timeDiff/constNeuronParams.membraneTau);
			float convTempValue = 0.0;    

			int tempId_x,tempId_y;

			tempId_x = my_addrx - min_x;
			tempId_y = my_addry - min_y;

			// make this texture cache access
			// convTempValue  = gpu_conv_template[params.objectId][tempId_y][tempId_x];
			int texPos = /*params.objectId*/0*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
		    convTempValue = tex1Dfetch (template_tex, texPos);
		    
			if (temp >  10 || (temp < 0)) {
				gpu_membranePotential[objId][my_addry][my_addrx] = convTempValue;		
			}
			else{
				float refValue = gpu_membranePotential[objId][my_addry][my_addrx];
				refValue = refValue*__expf(-temp) + convTempValue;
				//if(temp < 1)
				//	refValue = refValue*(1-temp) + convTempValue;
				//else
				//	refValue = convTempValue;
				gpu_lastTimeStamp[objId][my_addry][my_addrx]     = timeStamp;
				
				if (refValue > constNeuronParams.threshold)  {
					refValue = 0.0;
					int fireId = atomicAdd(numFiring, 1);
					atomicAdd(&totFiring,1);
					firedNeuronAddr[fireId] = (my_addry<<8)+ my_addrx;
				}
				else if (refValue < constNeuronParams.membranePotentialMin) 
					refValue = constNeuronParams.membranePotentialMin;
					
		
				// write back the calculated refValue 
				gpu_membranePotential[objId][my_addry][my_addrx] = refValue;
		   
		   } 
		}  // timeDiff calculation ends 
	} // if boundary calculation ends
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
convNN_singleSpikeKernel( cudaParameters_t params, int prevFired, int firingId)
{    
	int objId = 0;
	
    // Block index
    int bx = blockIdx.x; //blockIdx.x%4;
    int by = blockIdx.y; //blockIdx.x/4;
	int bz = blockIdx.z;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	// location of conv. blocks
//	int eventId = bz;
//	int subTemplateId_x = bx;
//	int subTemplateId_y = by;
	
	int* numFiring;
	
	if ( firingId )
		numFiring = &numFiring1;
	else
		numFiring = &numFiring0;

	if ( prevFired ) {
		if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.y == 0)) {
			if (firingId)
				numFiring0 = 0;
			else
				numFiring1 = 0;
		}
	}	
	
	if ( bz < params.len )  {

		unsigned long timeStamp  = params.timeStampV ; //*(params.timeStamp + eventId);
		
#if 1			
		unsigned int addr  = params.addrV;//*(params.addr + eventId);
		unsigned int addrx = (addr)&0xff;
		unsigned int addry = (addr>>8)&0xff;
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
		int max_y = addry + (MAX_TEMPLATE_SIZE/2);
		
		if (max_x >= MAX_X )
		   max_x = MAX_X - 1;
		
		if (max_y >= MAX_Y )
		   max_y = MAX_Y - 1;

		int align_min_y = min_y;
		int align_min_x;
		
		if ( min_x >= 0 )
			align_min_x = min_x - min_x%16;	
		else 
			align_min_x = 0;
			
		if ( min_y < 0 )
			align_min_y  = 0;	
#else			
	
		const int align_min_x = params.align_min_x;
		const int align_min_y = params.align_min_y;
		const int max_x = params.max_x;
		const int max_y= params.max_y;
		const int min_x = params.min_x;
		const int min_y = params.min_y;		
#endif		
		int3 globalId;
				
		globalId.x = tx + (bx * blockDim.x);
		globalId.y = ty + (by * blockDim.y);
		
		int my_addrx  = align_min_x + globalId.x;
		int my_addry  = align_min_y + globalId.y;
		
		if ( my_addrx >= 0 &&
			 my_addry >= 0 &&
			 my_addrx >= min_x &&
			 my_addry >= min_y &&
			 my_addrx <= max_x &&
			 my_addry <= max_y ) {			
				   
			int tempId_x = my_addrx - min_x;
			int tempId_y = my_addry - min_y;			
		
			unsigned long ltStamp = gpu_lastTimeStamp[objId][my_addry][my_addrx];
		
			// we are within logical boundary of frame
			//signed long long timeDiff = 0xFFFFFFFFLL&(timeStamp-ltStamp); 
		    unsigned long timeDiff = timeStamp-ltStamp;
		    //unsigned long tDiff = timeStamp - ltStamp;	
			//if ( tDiff & 0x80000000L)
			//	tDiff = 0;			
			//signed long timeDiff = (signed long)(-tDiff);
		    
			float temp = (float)(timeDiff/constNeuronParams.membraneTau);
			float convTempValue = 0.0;    

			// make this texture cache access
			// convTempValue  = gpu_conv_template[params.objectId][tempId_y][tempId_x];
			int texPos = params.objectId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
		    convTempValue = tex1Dfetch (template_tex, texPos);
		    
			if (temp >  10 || (temp < 0)) {
				gpu_membranePotential[objId][my_addry][my_addrx] = convTempValue;		
			}
			else{
				float refValue = gpu_membranePotential[objId][my_addry][my_addrx];
				refValue = refValue*__expf(-temp) + convTempValue;
				gpu_lastTimeStamp[objId][my_addry][my_addrx]     = timeStamp;
				
				if (refValue > constNeuronParams.threshold)  {
					refValue = 0.0;									
					int fireId = atomicAdd(numFiring, 1);
					atomicAdd(&totFiring,1);
					firedNeuronAddr[fireId] = (my_addry<<8)+ my_addrx;
				}
				else if (refValue < constNeuronParams.membranePotentialMin) 
					refValue = constNeuronParams.membranePotentialMin;
		
				// write back the calculated refValue 
				gpu_membranePotential[objId][my_addry][my_addrx] = refValue;		   
		   } 
		}  // timeDiff calculation ends 
	} // if boundary calculation ends
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
generateGlobalTemplateKernel2( cudaParameters_t params, int prevFired, int firingId)
{    
	// TODO: Temporarily we are only using one object map
	int objId = 0;
	
    // Block index
    int bx = blockIdx.x; //blockIdx.x%4;
    int by = blockIdx.y; //blockIdx.x/4;
	int bz = blockIdx.z;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	// location of conv. blocks
//	int eventId = bz;
//	int subTemplateId_x = bx;
//	int subTemplateId_y = by;
	
	int* numFiring;
	
	if ( firingId )
		numFiring = &numFiring1;
	else
		numFiring = &numFiring0;

	if ( prevFired ) {
		if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.y == 0)) {
			if (firingId)
				numFiring0 = 0;
			else
				numFiring1 = 0;
		}
	}	
	
	if ( bz < params.len )  {

		unsigned long timeStamp  = params.timeStampV ; //*(params.timeStamp + eventId);
		
#if 1			
		unsigned int addr  = params.addrV;//*(params.addr + eventId);
		unsigned int addrx = (addr)&0xff;
		unsigned int addry = (addr>>8)&0xff;
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
		int max_y = addry + (MAX_TEMPLATE_SIZE/2);
		
		if (max_x >= MAX_X )
		   max_x = MAX_X - 1;
		
		if (max_y >= MAX_Y )
		   max_y = MAX_Y - 1;

		int align_min_y = min_y;
		int align_min_x;
		
		if ( min_x >= 0 )
			align_min_x = min_x - min_x%16;	
		else 
			align_min_x = 0;
			
		if ( min_y < 0 )
			align_min_y  = 0;	
#else			
	
		const int align_min_x = params.align_min_x;
		const int align_min_y = params.align_min_y;
		const int max_x = params.max_x;
		const int max_y= params.max_y;
		const int min_x = params.min_x;
		const int min_y = params.min_y;		
#endif		
		int3 globalId;
				
		globalId.x = tx + (bx * blockDim.x);
		globalId.y = ty + (by * blockDim.y);
		
		int my_addrx  = align_min_x + globalId.x;
		int my_addry  = align_min_y + globalId.y;
		
		if ( my_addrx >= 0 &&
			 my_addry >= 0 &&
			 my_addrx >= min_x &&
			 my_addry >= min_y &&
			 my_addrx <= max_x &&
			 my_addry <= max_y ) {			
				   
			int tempId_x = my_addrx - min_x;
			int tempId_y = my_addry - min_y;			
		
			unsigned long ltStamp = gpu_lastTimeStamp[objId][my_addry][my_addrx];
		
			// we are within logical boundary of frame
			//signed long long timeDiff = 0xFFFFFFFFLL&(timeStamp-ltStamp); 
		    unsigned long timeDiff = timeStamp-ltStamp;
		    //unsigned long tDiff = timeStamp - ltStamp;	
			//if ( tDiff & 0x80000000L)
			//	tDiff = 0;			
			//signed long timeDiff = (signed long)(-tDiff);
		    
			float temp = (float)(timeDiff/constNeuronParams.membraneTau);
			float convTempValue = 0.0;    

			// make this texture cache access
			// convTempValue  = gpu_conv_template[params.objectId][tempId_y][tempId_x];
			int texPos = params.objectId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
		    convTempValue = tex1Dfetch (template_tex, texPos);
		    
			if (temp >  10 || (temp < 0)) {
				gpu_membranePotential[objId][my_addry][my_addrx] = convTempValue;		
			}
			else{
				float refValue = gpu_membranePotential[objId][my_addry][my_addrx];
				refValue = refValue*__expf(-temp) + convTempValue;
				gpu_lastTimeStamp[objId][my_addry][my_addrx]     = timeStamp;
				
				if (refValue > constNeuronParams.threshold)  {
					refValue = 0.0;									
					int fireId = atomicAdd(numFiring, 1);
					atomicAdd(&totFiring,1);
					firedNeuronAddr[fireId] = (my_addry<<8)+ my_addrx;
				}
				else if (refValue < constNeuronParams.membranePotentialMin) 
					refValue = constNeuronParams.membranePotentialMin;
		
				// write back the calculated refValue 
				gpu_membranePotential[objId][my_addry][my_addrx] = refValue;		   
		   } 
		}  // timeDiff calculation ends 
	} // if boundary calculation ends
}

#define SPIKE_BUFFER_LEN 128

__device__ unsigned long gpu_blockLastFiring[MAX_NUM_BLOCKS][MAX_NUM_BLOCKS];
__device__ float gpu_iNeuronPotentialBlock[MAX_NUM_BLOCKS];
__device__ int spikeInfo[GPU_MAX_SPIKE_PACKETS];

__global__ void
convNN_multiSpikeKernel( cudaParameters_t params, int prevFired, int firingId)
{
	// TODO: Temporarily we are only using one object map
	int objId = 0;
	
	__shared__ volatile int sh_potential;
	__shared__ volatile float sh_decayFactor;      
	__shared__ volatile int sh_numInpSpikes;   
   
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int my_addrx = blockIdx.x*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = blockIdx.y*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;   
    int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block 	

	int* numFiring;
	
	if ( firingId )
		numFiring = &numFiring1;
	else
		numFiring = &numFiring0;

	if ( prevFired && ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.y == 0))) {
		if (firingId)
			numFiring0 = 0;
		else
			numFiring1 = 0;			
	}
			
   __syncthreads();
  
   float refValue  = gpu_membranePotential[objId][my_addry][my_addrx];    
   unsigned long timeStamp = params.timeStampV;;   
   
   if(threadIdx.x == 0) {
		unsigned long ltStamp = gpu_lastTimeStamp[objId][my_addry][my_addrx];		
		unsigned long timeDiff = timeStamp-ltStamp;	    
		float temp = (float)(timeDiff/constNeuronParams.membraneTau);
		sh_decayFactor = __expf(-temp);		
   }
   __syncthreads();   

    /* get the reference potential value */
    refValue = refValue*sh_decayFactor;
    
    //bool neuronSelected = false;
    __shared__ volatile unsigned int sh_spike_addr[SPIKE_BUFFER_LEN];        
	int cnt=SPIKE_BUFFER_LEN;
	for(int i=0; i < params.len;) {
		
		if(cnt==SPIKE_BUFFER_LEN)	{
			// all thread with address less then spike buffer length will fetch data from memory				
			if(my_localId < SPIKE_BUFFER_LEN)	
				sh_spike_addr[my_localId] = spikeInfo[i+my_localId]; //params.addrV;//*(params.addr + eventId);
			__syncthreads();
			cnt=0;				
		}	    			
				
		unsigned int addrx = (sh_spike_addr[cnt])&0xff;
		unsigned int addry = (sh_spike_addr[cnt]>>8)&0xff;
		cnt++;
		i++;

		/* region of template that is valid */
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
		int max_y = addry + (MAX_TEMPLATE_SIZE/2);

		if (max_x >= MAX_X )
		   max_x = MAX_X - 1;

		if (max_y >= MAX_Y )
		   max_y = MAX_Y - 1;

		if ( my_addrx >= min_x &&
			 my_addry >= min_y &&
			 my_addrx <= max_x &&
			 my_addry <= max_y ) {
			
	   		int tempId_x = my_addrx - min_x;
			int tempId_y = my_addry - min_y;
			
			/* we can read the template and get a valid data */
			int texPos = params.objectId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
			refValue = refValue + tex1Dfetch (template_tex, texPos);
			//neuronSelected = true;
		}

		if (refValue > constNeuronParams.threshold)  {
			refValue = 0.0;									
			int fireId = atomicAdd(numFiring, 1);
			atomicAdd(&totFiring,1);
			firedNeuronAddr[fireId] = (my_addry<<8)+ my_addrx;
		}
		else if (refValue < constNeuronParams.membranePotentialMin) 
			refValue = constNeuronParams.membranePotentialMin;

#if LOOP_UNROLL_2
		
		if(cnt >= params.len)
			break;
			
		unsigned int addrx1 = (sh_spike_addr[cnt+1])&0xff;
		unsigned int addry1 = (sh_spike_addr[cnt+1]>>8)&0xff;
		cnt++;
		i++;

		/* region of template that is valid */
		int min_x1 = addrx1 - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y1 = addry1 - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x1 = addrx1 + (MAX_TEMPLATE_SIZE/2);
		int max_y1 = addry1 + (MAX_TEMPLATE_SIZE/2);

		if (max_x1 >= MAX_X )
		   max_x1 = MAX_X - 1;

		if (max_y1 >= MAX_Y )
		   max_y1 = MAX_Y - 1;

		if ( my_addrx >= min_x1 &&
			 my_addry >= min_y1 &&
			 my_addrx <= max_x1 &&
			 my_addry <= max_y1 ) {
			
	   		int tempId_x = my_addrx - min_x1;
			int tempId_y = my_addry - min_y1;
			
			/* we can read the template and get a valid data */
			int texPos = params.objectId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
			refValue = refValue + tex1Dfetch (template_tex, texPos);
			//neuronSelected = true;
		}

		if (refValue > constNeuronParams.threshold)  {
			refValue = 0.0;									
			int fireId = atomicAdd(numFiring, 1);
			atomicAdd(&totFiring,1);
			firedNeuronAddr[fireId] = (my_addry<<8)+ my_addrx;
		}
		else if (refValue < constNeuronParams.membranePotentialMin) 
			refValue = constNeuronParams.membranePotentialMin;			
#endif			
	}
	

    if(threadIdx.x == 0)
    	gpu_lastTimeStamp[objId][my_addry][my_addrx] = timeStamp;		

	// write back the calculated refValue 
	//if (neuronSelected)
	gpu_membranePotential[objId][my_addry][my_addrx] = refValue;
}

__device__ int tmp[MAX_NUM_OBJECT][1000];
__global__ void
convNN_multiObjectmultiSpikeKernel( cudaParameters_t params, int prevFired, int firingId, int callCount)
{
	__shared__ volatile int sh_potential;
	__shared__ volatile float sh_decayFactor;      
	__shared__ volatile int sh_numInpSpikes;   
   
	/* blockIdx.x,blockIdx.y can range from 0-31 */
	int bx = blockIdx.x;
	//TODO: This code is specific to a image of size 128, with 8 blocks
	//each operating 16x16 pixel array.
	//We encode the object dimension in blockId itself.last 3 bit denotes
	//block number, the remaining bits denote the object number
	int by = (blockIdx.y&0x7);
	int bz = (blockIdx.y>>3);
	int my_addrx = bx*MAX_SUB_TEMPLATE_SIZE_X + threadIdx.x;
	int my_addry = by*MAX_SUB_TEMPLATE_SIZE_Y + threadIdx.y;  	
	int my_localId = threadIdx.y*blockDim.x+threadIdx.x; // unique local id within a block 	
	int objId = bz;
		 
	int* numFiring;
	
	if ( firingId )
		numFiring = &numFiring1MO[objId];
	else
		numFiring = &numFiring0MO[objId];
	
	if ((bx == 0) && (by == 0) && (threadIdx.x == 0) && (threadIdx.y == 0)) {		
		if( prevFired ) {
			if (firingId)
				numFiring0MO[objId] = 0;
			else
				numFiring1MO[objId] = 0;			
		}
	}
			
	__syncthreads();
 
	//for each object we have a separate array of membrane potential. This can
	//can be called as neuron area0. Each area is sensitive to one specific object
	//template. Currently we evaluate the given spike for all the area, 
	//but return the result corresponding to only the area0. 
	//TODO: future work would include picking up the peak response from
	//one of the area using a winner-take-all network. so that we can
	//select the peak responding area and associate it with the object
	float refValue = gpu_membranePotential[objId][my_addry][my_addrx];
	unsigned long timeStamp = params.timeStampV;

	if(threadIdx.x == 0) {
		unsigned long ltStamp = gpu_lastTimeStamp[objId][my_addry][my_addrx];
		unsigned long timeDiff = timeStamp-ltStamp;
		float temp = (float)(timeDiff/constNeuronParams.membraneTau);
		sh_decayFactor = __expf(-temp);
	}
	__syncthreads();

	/* get the reference potential value */
	refValue = refValue*sh_decayFactor;
    
	//bool neuronSelected = false;
	__shared__ volatile unsigned int sh_spike_addr[SPIKE_BUFFER_LEN];        
	int cnt=SPIKE_BUFFER_LEN;
	for(int i=0; i < params.len; i+=1) {
	
		if(cnt==SPIKE_BUFFER_LEN)	{
			// all thread with address less then spike buffer length will fetch data from memory				
			if(my_localId < SPIKE_BUFFER_LEN)	
				sh_spike_addr[my_localId] = spikeInfo[i+my_localId]; //params.addrV;//*(params.addr + eventId);
			__syncthreads();
			cnt=0;				
		}	    									
		unsigned int addrx = (sh_spike_addr[cnt])&0xff;
		unsigned int addry = (sh_spike_addr[cnt]>>8)&0xff;
		cnt++;
		
		/* region of template that is valid */
		int min_x = addrx - (MAX_TEMPLATE_SIZE/2) + 1;
		int min_y = addry - (MAX_TEMPLATE_SIZE/2) + 1;
		int max_x = addrx + (MAX_TEMPLATE_SIZE/2);
		int max_y = addry + (MAX_TEMPLATE_SIZE/2);

		if (max_x >= MAX_X )
		   max_x = MAX_X - 1;

		if (max_y >= MAX_Y )
		   max_y = MAX_Y - 1;

		if ( my_addrx >= min_x &&
			 my_addry >= min_y &&
			 my_addrx <= max_x &&
			 my_addry <= max_y ) {
			
			int tempId_x = my_addrx - min_x;
			int tempId_y = my_addry - min_y;
			
			/* we can read the template and get a valid data */
			int texPos = objId*MAX_TEMPLATE_SIZE*MAX_TEMPLATE_SIZE + tempId_y*MAX_TEMPLATE_SIZE + tempId_x;
			refValue = refValue + tex1Dfetch (template_tex, texPos);
			//neuronSelected = true;
		}
		
		if (refValue > constNeuronParams.threshold)  {
			refValue = 0.0;												
			// TODO: currently we track all object but put them in only
			// one queue. Update to include the object id along with spiking location			
			int fireId = atomicAdd(numFiring, 1);
			atomicAdd(&totFiring,1);			
			firedNeuronAddr[objId*MAX_FIRING+fireId] = (my_addry<<8)+ my_addrx;			
		}
		else if (refValue < constNeuronParams.membranePotentialMin) 
			refValue = constNeuronParams.membranePotentialMin;
	}

	if(threadIdx.x == 0)
		gpu_lastTimeStamp[objId][my_addry][my_addrx] = timeStamp;		

	// write back the calculated refValue 
	gpu_membranePotential[objId][my_addry][my_addrx] = refValue;
	
}
#endif // #ifndef _TEMPLATE_KERNEL_H_