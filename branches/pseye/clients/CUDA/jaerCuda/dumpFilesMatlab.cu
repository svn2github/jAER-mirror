#include "hip/hip_runtime.h"
// includes, project
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>

#include "config.h"

extern "C"	{
	void dumpTemplate(FILE* fp, char* fstr);
	void printResults(FILE* fpLog);
	void showMembranePotential(unsigned int* spikeAddr, int spikeCnt);
}

// Function dumps the template parameters into a file
// pointed by fp. The file can be executed in matlab
void dumpTemplate(FILE* fp, char* fstr)
{
	if(fp==NULL)
		return;

	printf( "Dumping %d templates to %s\n", num_object, fstr);
	fflush(stdout); // for jaer to print this
	for(int i = 0; i < num_object; i++){		
		fprintf( fp, " template%d = [ ", i);
		for(int j=0; j < MAX_TEMPLATE_SIZE; j++) {
			for(int k=0; k < MAX_TEMPLATE_SIZE; k++) {
				fprintf( fp, " %f ", conv_template[i][j][k]);
			}
			fprintf(fp, "; \n");
		}
		fprintf(fp , " ];\nfigure;imagesc(template%d);\n\n",i);		
	}
	fflush(fp);	

}

int countMem = 0;

// only for debug, writes the membrane potentials to a file
void showMembranePotential(unsigned int* spikeAddr=NULL, int spikeCnt=0)
{		
		if((countMem >= RECORD_START && countMem <= RECORD_END))
		{
			if(runCuda)
				hipMemcpyFromSymbol(membranePotential, HIP_SYMBOL("gpu_membranePotential"), sizeof(membranePotential), 0, hipMemcpyDeviceToHost);

			char fname[100];
			sprintf(fname, "mem_pot%d.m", countMem);
			FILE* fpDumpPot;	
			fpDumpPot = fopen(fname, "w");
	
			for(int k = 0; k < num_object; k++){
				fprintf( fpDumpPot, " memPot[%d] = [ ", k);
				for(int i=0; i < MAX_Y; i++) {
					for(int j=0; j < MAX_X; j++) {
						fprintf( fpDumpPot, " %f ", membranePotential[k][i][j]);
					}
					fprintf(fpDumpPot, "; \n");
				}
				fprintf( fpDumpPot, "];\n\n");
			}

			fclose(fpDumpPot);

			if(spikeAddr != NULL) {
				char fname[100];
				sprintf(fname, "inpSpike%d.m", countMem);
				FILE* fpDumpPot;
				fpDumpPot = fopen(fname, "w");
				fprintf( fpDumpPot, " inpSpike = [ " );
				for(int j=0; j < spikeCnt; j++) {					
					fprintf( fpDumpPot, " %u ", spikeAddr[j]);
				}
				fprintf(fpDumpPot , " ]; " );
				fclose(fpDumpPot);	
			}		
		}	
		
		countMem++;		

}


void printResults(FILE* fpLog)
{
	int tot_fired = 0;
	if(!runCuda) {
		extern int cpu_totFiring;
		extern int cpu_totFiringMO[MAX_NUM_TEMPLATE];
		printf(" Number of fired neurons is %d\n", cpu_totFiring);	
		printf(" Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);					
		fprintf(fpLog, " Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);	
		fprintf(fpLog, " Number of fired neurons is %d\n", tot_fired);
		for(int i=0; i < num_object; i++) {
			printf(" Total firing in Array %d => %d\n", i, cpu_totFiringMO[i]);
			fprintf(fpLog, " Total firing in Array %d => %d\n", i, cpu_totFiringMO[i]);		
		}
	}
	else {
		printf("Kernel 1 called %d times\n", callCount);
		printf(" Total number of spikes computed : %d\n", tot_filteredSpikes);
		for(int i = 0; i < num_object; i++)
			tot_fired = tot_fired + tot_fired_MO[i]; 
		printf(" Number of fired neurons is %d\n", tot_fired);	
		printf(" Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);			
		fprintf(fpLog, "Kernel 1 called %d times\n", callCount);
		fprintf(fpLog, " Total number of spikes computed : %d\n", tot_filteredSpikes);
		fprintf(fpLog, " Number of fired neurons is %d\n", tot_fired);
		fprintf(fpLog, " Template size is %dx%d\n", MAX_TEMPLATE_SIZE, MAX_TEMPLATE_SIZE);

		for(int i=0; i < num_object; i++) {
			printf(" Total firing in Array %d => %d\n", i, tot_fired_MO[i]);
			fprintf(fpLog, " Total firing in Array %d => %d\n", i, tot_fired_MO[i]);													
		}
	}
	
	int tot=0;
	int minLen=1000;
	int mini=0;
	int maxLen=-1;
	int maxi=0;
	int cnt=(callCount>PARAM_LEN_SIZE)?PARAM_LEN_SIZE:callCount;
	for(int i=0; i < cnt; i++) {
	  tot+= paramLenArr[i];
	  if (paramLenArr[i] < minLen) {minLen =paramLenArr[i];  mini=i; }
	  if (paramLenArr[i] > maxLen) {maxLen = paramLenArr[i]; maxi=i; }
	}
	if(runCuda) {
		printf(" Spike Distribution Per Kernel Call: \nmean(%f), min (i=%d, val=%d), max(i=%d,val=%d)\n", tot*1.0/cnt, mini, minLen, maxi,maxLen);
		fprintf(fpLog, " Spike Distribution Per Kernel Call: \nmean(%f), min (i=%d, val=%d), max(i=%d,val=%d)\n", tot*1.0/cnt, mini, minLen, maxi,maxLen);
	}

	if(runCuda) {
		printf( " Total Object scanned : %d\n", num_object);
		printf( " Total firing from Inhibition Neuron : %d\n", inhFireCnt);
		printf( " Total firing is equal to %d\n", tot_fired);	
		printf( " Average firing is equal to %f\n", tot_fired*1.0/callCount);
		printf( "\n\nAvg. GPU Processing time per spike: %f (ms)\n", accTimer/(tot_filteredSpikes));
		printf( "\n\nTotal GPU Processing time : %f (ms)\n", accTimer);
		fprintf( fpLog,  " Total Object scanned : %d\n", num_object);
		fprintf( fpLog,  " Total firing from Inhibition Neuron : %d\n", inhFireCnt);
		fprintf( fpLog,  " Total firing is equal to %d\n", tot_fired);
		fprintf( fpLog,  "\n\nAvg. GPU Processing time per spike: %f (ms)\n", accTimer/(tot_filteredSpikes));
		fprintf( fpLog,  "\n\nTotal GPU Processing time : %f (ms)\n", accTimer);	
	}
	else {	
		extern int cpu_totFiring;
		extern int iNeuronFiringCnt;
		extern int iNeuronCallingCnt;    
		printf( " Total Object scanned : %d\n", num_object);
		printf("  INeuron Grouping Impact, Calls = %d, Firing = %d\n", iNeuronCallingCnt, iNeuronFiringCnt);
		printf( " Total firing is equal to %d\n", cpu_totFiring);
		printf( "\n\nCPU Processing time per spike: %f (ms)\n",  accTimer/(tot_filteredSpikes));
		printf( "\n\nTotal CPU Processing time : %f (ms)\n", accTimer);
		fprintf( fpLog,  " Total Object scanned : %d\n", num_object);
		fprintf( fpLog,  " Total firing from Inhibition Neuron : %d\n", iNeuronFiringCnt);
		fprintf( fpLog,  " Total firing is equal to %d\n", cpu_totFiring);
		fprintf( fpLog,  "\n\nCPU Processing time per spike: %f (ms)\n",  accTimer/(tot_filteredSpikes));
		fprintf( fpLog, "\n\nTotal CPU Processing time : %f (ms)\n", accTimer);	
	}

	fflush(stdout);  // so jaer gets it
}

